#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <stdio.h>
#include <iostream>
#include <cmath>

#include "float32.h"

#define IN_DATA_BYTES (IN_SIZE*sizeof(dtype))
#define OUT_DATA_BYTES (OUT_SIZE*sizeof(dtype))

//function to print out error message from cuDNN calls
#define checkCUDNN(exp) \
  { \
    hipdnnStatus_t status = (exp); \
    if(status != HIPDNN_STATUS_SUCCESS) { \
      std::cerr << "Error on line " << __LINE__ << ": " \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE); \
    } \
  } 

int main() {
  hipdnnHandle_t cudnn;
  checkCUDNN(hipdnnCreate(&cudnn));

  hipdnnPoolingDescriptor_t pooling_desc;
  //create descriptor handle
  checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_desc));
  //initialize descriptor
  checkCUDNN(hipdnnSetPooling2dDescriptor(pooling_desc,            //descriptor handle
                                         HIPDNN_POOLING_MAX,       //mode - max pooling
                                         HIPDNN_NOT_PROPAGATE_NAN, //NaN propagation mode
                                         3,                       //window height
                                         3,                       //window width
                                         0,                       //vertical padding
                                         0,                       //horizontal padding
                                         1,                       //vertical stride
                                         1));                     //horizontal stride
  
  hipdnnTensorDescriptor_t in_desc;
  //create input data tensor descriptor
  checkCUDNN(hipdnnCreateTensorDescriptor(&in_desc));
  //initialize input data descriptor 
  checkCUDNN(hipdnnSetTensor4dDescriptor(in_desc,                  //descriptor handle
                                        HIPDNN_TENSOR_NCHW,        //data format
                                        CUDNN_DTYPE,              //data type (precision)
                                        2,                        //number of images
                                        2,                        //number of channels
                                        10,                       //data height 
                                        10));                     //data width

  hipdnnTensorDescriptor_t out_desc;
  //create output data tensor descriptor
  checkCUDNN(hipdnnCreateTensorDescriptor(&out_desc));
  //initialize output data descriptor
  checkCUDNN(hipdnnSetTensor4dDescriptor(out_desc,                 //descriptor handle
                                        HIPDNN_TENSOR_NCHW,        //data format
                                        CUDNN_DTYPE,              //data type (precision)
                                        2,                        //number of images
                                        2,                        //number of channels
                                        8,                        //data height
                                        8));                      //data width

  stype alpha = 1.0f;
  stype beta = 0.0f;
  //GPU data pointers
  dtype *in_data, *out_data;
  //allocate arrays on GPU
  hipMalloc(&in_data,IN_DATA_BYTES);
  hipMalloc(&out_data,OUT_DATA_BYTES);
  //copy input data to GPU array
  hipMemcpy(in_data,input,IN_DATA_BYTES,hipMemcpyHostToDevice);
  //initize output data on GPU
  hipMemset(out_data,0,OUT_DATA_BYTES);

  //Call pooling operator
  checkCUDNN(hipdnnPoolingForward(cudnn,         //cuDNN context handle
                                 pooling_desc,  //pooling descriptor handle
                                 &alpha,        //alpha scaling factor
                                 in_desc,       //input tensor descriptor
                                 in_data,       //input data pointer to GPU memory
                                 &beta,         //beta scaling factor
                                 out_desc,      //output tensor descriptor
                                 out_data));    //output data pointer from GPU memory

  //allocate array on CPU for output tensor data
  dtype *result = (dtype*)malloc(OUT_DATA_BYTES);
  //copy output data from GPU
  hipMemcpy(result,out_data,OUT_DATA_BYTES,hipMemcpyDeviceToHost);

  //loop over and check that the forward pass outputs match expected results (exactly)
  int err = 0;
  for(int i=0; i<OUT_SIZE; i++) {
    if(result[i] != output[i]) {
      std::cout << "Error! Expected " << output[i] << " got " << result[i] << " for idx " << i <<std::endl;
      err++;
    }
  }

  std::cout << "Forward finished with " << err << " errors" << std::endl;

  dtype *in_grad;
  //allocate array on GPU for gradient
  hipMalloc(&in_grad,IN_DATA_BYTES);
  //initialize output array 
  hipMemset(in_grad,0,IN_DATA_BYTES);

  //call pooling operator to compute gradient
  checkCUDNN(hipdnnPoolingBackward(cudnn,        //cuDNN context handle
                                  pooling_desc, //pooling descriptor handle
                                  &alpha,       //alpha scaling factor
                                  out_desc,     //output tensor descriptor
                                  out_data,     //output tensor pointer to GPU memory
                                  out_desc,     //differential tensor descriptor
                                  out_data,     //differential tensor pointer to GPU memory
                                  in_desc,      //input tensor descriptor
                                  in_data,      //input tensor pointer to GPU memory
                                  &beta,        //beta scaling factor
                                  in_desc,      //gradient tensor descriptor
                                  in_grad));    //gradient tensor pointer to GPU memory

  //allocate array on CPU for gradient tensor data
  dtype *grad = (dtype*)malloc(IN_DATA_BYTES);
  //copy gradient data from GPU
  hipMemcpy(grad,in_grad,IN_DATA_BYTES,hipMemcpyDeviceToHost);

  //loop over and check that the forward pass outputs match expected results (within tolerance)
  err = 0;
  for(int i=0; i<IN_SIZE; i++) {
    double diff = std::abs(gradient[i] - grad[i]);
    if(diff > TOL) {
      std::cout << "Error! Expected " << gradient[i] << " got " << grad[i] << " for idx " << i << " diff: " << diff <<std::endl;
      err++;
    }
  }

  std::cout << "Backward finished with " << err << " errors" << std::endl;

  //free CPU arrays
  free(result);
  free(grad);

  //free GPU arrays
  hipFree(in_data);
  hipFree(in_grad);
  hipFree(out_data);

  //free cuDNN descriptors
  hipdnnDestroyTensorDescriptor(in_desc);
  hipdnnDestroyTensorDescriptor(out_desc);
  hipdnnDestroyPoolingDescriptor(pooling_desc);
  hipdnnDestroy(cudnn);
  
  return 0;
}
