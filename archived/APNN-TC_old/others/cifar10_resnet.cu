#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// File: cifar10_resnet.cu
// ResNet-14 BNN inference source file for CIFAR10. 
// ---------------------------------------------------------------------------
// See our arXiv paper for detail: https://arxiv.org/abs/2006.16578
// Ang Li, Scientist, Pacific Northwest National Laboratory(PNNL), U.S.
// Homepage: http://www.angliphd.com
// GitHub repo: http://www.github.com/pnnl/TCBNN
// PNNL-IPID: 31925-E, ECCN: EAR99, IR: PNNL-SA-152850
// BSD Lincese.
// Richland, 99352, WA, USA. June-30-2020.
// ---------------------------------------------------------------------------

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <iostream>
#include <string>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <fstream>
#include <vector>
#include "utility.h"
#include "param.h"
#include "kernel.cuh"
#include "data.h"

using namespace cooperative_groups;
using namespace std;

#ifdef NEWFMT
__global__ void resnet128(
        InConv128LayerParam* bconv1, 
        Conv128LayerParam* l1b1c1, 
        Conv128LayerParam* l1b1c2,
        Conv128LayerParam* l1b2c1, 
        Conv128LayerParam* l1b2c2,
        Conv128LayerParam* l2b1c1, 
        Conv128LayerParam* l2b1c2,
        Conv128LayerParam* l2b2c1, 
        Conv128LayerParam* l2b2c2,
        Conv128LayerParam* l3b1c1, 
        Conv128LayerParam* l3b1c2,
        Conv128LayerParam* l3b2c1, 
        Conv128LayerParam* l3b2c2,
        Fc128LayerParam* bfc1, 
        Out128LayerParam* bout)
{
    grid_group grid = this_grid();
    //SET_KERNEL_TIMER;
    //========= Conv1 ============
    InConv128LayerFMT(bconv1);
    grid.sync();
    //TICK_KERNEL_TIMER(bconv1);
    //========= L1B1 ============
    Conv128LayerFMT(l1b1c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l1b1c1);
    Conv128LayerFMT(l1b1c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l1b1c2);
    //========= L1B2 ============
    Conv128LayerFMT(l1b2c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l1b2c1);
    Conv128LayerFMT(l1b2c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l1b2c2);
    //========= L2B1 ============
    Conv128LayerFMT(l2b1c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l2b1c1);
    Conv128LayerFMT(l2b1c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l2b1c2);
    //========= L2B2 ============
    Conv128LayerFMT(l2b2c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l2b2c1);
    Conv128LayerFMT(l2b2c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l2b2c2);
    //========= L3B1 ============
    Conv128LayerFMT(l3b1c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l3b1c1);
    Conv128LayerFMT(l3b1c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l3b1c2);
    //========= L3B2 ============
    Conv128LayerFMT(l3b2c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l3b2c1);
    Conv128LayerFMT(l3b2c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l3b2c2);
    //========= Fc1 ============
    Fc128LayerFMT(bfc1);
    grid.sync();
    //TICK_KERNEL_TIMER(bfc1);
    ////========== Output ===========
    Out128LayerFMT(bout);
    //grid.sync();
    //TICK_KERNEL_TIMER(bout);
}

#else
__global__ void resnet128(
        InConv128LayerParam* bconv1, 
        Conv128LayerParam* l1b1c1, 
        Conv128LayerParam* l1b1c2,
        Conv128LayerParam* l1b2c1, 
        Conv128LayerParam* l1b2c2,
        Conv128LayerParam* l2b1c1, 
        Conv128LayerParam* l2b1c2,
        Conv128LayerParam* l2b2c1, 
        Conv128LayerParam* l2b2c2,
        Conv128LayerParam* l3b1c1, 
        Conv128LayerParam* l3b1c2,
        Conv128LayerParam* l3b2c1, 
        Conv128LayerParam* l3b2c2,
        Fc128LayerParam* bfc1, 
        Out128LayerParam* bout)
{
    grid_group grid = this_grid();
    //========= Conv1 ============
    InConv128Layer(bconv1);
    grid.sync();
    //========= L1B1 ============
    Conv128Layer(l1b1c1);
    grid.sync();
    Conv128Layer(l1b1c2);
    grid.sync();
    //========= L1B2 ============
    Conv128Layer(l1b2c1);
    grid.sync();
    Conv128Layer(l1b2c2);
    grid.sync();
    //========= L2B1 ============
    Conv128Layer(l2b1c1);
    grid.sync();
    Conv128Layer(l2b1c2);
    grid.sync();
    //========= L2B2 ============
    Conv128Layer(l2b2c1);
    grid.sync();
    Conv128Layer(l2b2c2);
    grid.sync();
    //========= L3B1 ============
    Conv128Layer(l3b1c1);
    grid.sync();
    Conv128Layer(l3b1c2);
    grid.sync();
    //========= L3B2 ============
    Conv128Layer(l3b2c1);
    grid.sync();
    Conv128Layer(l3b2c2);
    grid.sync();
    //========= Fc1 ============
    Fc128Layer(bfc1);
    grid.sync();
    ////========== Output ===========
    Out128Layer(bout);
}
#endif


int main()
{
    int dev = 0;
    hipSetDevice(dev);
    const unsigned batch = 512;
    const unsigned output_size = 10;
    const unsigned image_height = 32;
    const unsigned image_width = 32;
    const unsigned image_channel = 3;

    //=============== Get Input and Label =================
    float* images = (float*)malloc(batch*image_height*image_width*image_channel*sizeof(float));
    unsigned* image_labels = (unsigned*)malloc(batch*sizeof(unsigned));
    string cifar10_dir = "/home/lian599/data/cifar10c/test_batch.bin";
    read_CIFAR10_normalized(cifar10_dir, images, image_labels, batch);

    //================ Get Weight =================
    FILE* config_file = fopen("./resnet_cifar10.csv","r");

    //================ Set Network =================
    //Layer-0
    InConv128LayerParam* bconv1 = new InConv128LayerParam("Conv1", image_height, image_width, 
            3, 3, 3, 128, batch,1,1,true,1,1,true);//save residual 
    InConv128LayerParam* bconv1_gpu = bconv1->initialize(images, config_file);
    //Layer-1, basic-block-1, conv1
    Conv128LayerParam* l1b1c1 = new Conv128LayerParam("L1B1C1", bconv1->output_height, 
            bconv1->output_width, 3, 3, 128, 128, batch);
    Conv128LayerParam* l1b1c1_gpu = l1b1c1->initialize(config_file, bconv1->get_output_gpu());
    
    //Layer-1, basic-block-1, conv2
    Conv128LayerParam* l1b1c2 = new Conv128LayerParam("L1B1C2", l1b1c1->output_height, 
            l1b1c1->output_width, 3, 3, 128, 128, batch,1,1,true,1,1,false,true,true,128);
    Conv128LayerParam* l1b1c2_gpu = l1b1c2->initialize(config_file, l1b1c1->get_output_gpu(),
            bconv1->get_output_residual_gpu());

    //Layer-1, basic-block-2, conv1
    Conv128LayerParam* l1b2c1 = new Conv128LayerParam("L1B2C1", l1b1c2->output_height, 
            l1b1c2->output_width, 3, 3, 128, 128, batch);
    Conv128LayerParam* l1b2c1_gpu = l1b2c1->initialize(config_file, l1b1c2->get_output_gpu());
    //Layer-1, basic-block-2, conv2
    Conv128LayerParam* l1b2c2 = new Conv128LayerParam("L1B2C2", l1b2c1->output_height, 
            l1b2c1->output_width, 3, 3, 128, 128, batch,1,1,true,1,1,false,true,true,128);
    Conv128LayerParam* l1b2c2_gpu = l1b2c2->initialize(config_file, l1b2c1->get_output_gpu(),
            l1b1c2->get_output_residual_gpu());

    //Layer-2, basic-block-1, conv1
    Conv128LayerParam* l2b1c1 = new Conv128LayerParam("L2B1C1", l1b2c2->output_height, 
            l1b2c2->output_width, 3, 3, 128, 256, batch, 2, 2);
    Conv128LayerParam* l2b1c1_gpu = l2b1c1->initialize(config_file, l1b2c2->get_output_gpu());
    //Layer-2, basic-block-1, conv2
    Conv128LayerParam* l2b1c2 = new Conv128LayerParam("L2B1C2", l2b1c1->output_height, 
            l2b1c1->output_width, 3, 3, 256, 256, batch,1,1,true,1,1,false,true,true,128,true);
    Conv128LayerParam* l2b1c2_gpu = l2b1c2->initialize(config_file, l2b1c1->get_output_gpu(),
            l1b2c2->get_output_residual_gpu());

    //Layer-2, basic-block-2, conv1
    Conv128LayerParam* l2b2c1 = new Conv128LayerParam("L2B2C1", l2b1c2->output_height, 
            l2b1c2->output_width, 3, 3, 256, 256, batch);
    Conv128LayerParam* l2b2c1_gpu = l2b2c1->initialize(config_file, l2b1c2->get_output_gpu());

    //Layer-2, basic-block-2, conv2
    Conv128LayerParam* l2b2c2 = new Conv128LayerParam("L2B2C2", l2b2c1->output_height, 
            l2b2c1->output_width, 3, 3, 256, 256, batch,1,1,true,1,1,false,true,true,256);
    Conv128LayerParam* l2b2c2_gpu = l2b2c2->initialize(config_file, l2b2c1->get_output_gpu(),
            l2b1c2->get_output_residual_gpu());
    //Layer-3, basic-block-1, conv1
    Conv128LayerParam* l3b1c1 = new Conv128LayerParam("L3B1C1", l2b2c2->output_height, 
            l2b2c2->output_width, 3, 3, 256, 512, batch, 2, 2);
    Conv128LayerParam* l3b1c1_gpu = l3b1c1->initialize(config_file, l2b2c2->get_output_gpu());

    //Layer-3, basic-block-1, conv2
    Conv128LayerParam* l3b1c2 = new Conv128LayerParam("L3B1C2", l3b1c1->output_height, 
            l3b1c1->output_width, 3, 3, 512, 512, batch,1,1,true,1,1,false,true,true,256,true);
    Conv128LayerParam* l3b1c2_gpu = l3b1c2->initialize(config_file, l3b1c1->get_output_gpu(),
            l2b2c2->get_output_residual_gpu());

    //Layer-3, basic-block-2, conv1
    Conv128LayerParam* l3b2c1 = new Conv128LayerParam("L3B2C1", l3b1c2->output_height, 
            l3b1c2->output_width, 3, 3, 512, 512, batch, 1, 1);
    Conv128LayerParam* l3b2c1_gpu = l3b2c1->initialize(config_file, l3b1c2->get_output_gpu());

    //Layer-3, basic-block-2, conv2
    Conv128LayerParam* l3b2c2 = new Conv128LayerParam("L3B2C2", l3b2c1->output_height, 
            l3b2c1->output_width, 3, 3, 512, 512, batch,1,1,true,1,1,true,false,true,512);
    Conv128LayerParam* l3b2c2_gpu = l3b2c2->initialize(config_file, l3b2c1->get_output_gpu(),
            l3b1c2->get_output_residual_gpu());
    //Layer-4
    Fc128LayerParam* bfc1 = new Fc128LayerParam("Fc1", batch, (l3b2c2->output_height)
            *(l3b2c2->output_width)*512, 512); 
    Fc128LayerParam* bfc1_gpu = bfc1->initialize(config_file, l3b2c2->get_output_gpu());
    //Out Layer
    Out128LayerParam* bout = new Out128LayerParam("Fout", batch, 512, output_size);
    Out128LayerParam* bout_gpu = bout->initialize(config_file, bfc1->get_output_gpu());  

    //================ Setup Kernel =================
    int numThreads = 1024;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int numBlocksPerSm;
    int shared_memory = 512*sizeof(int)*32;
    hipFuncSetAttribute(reinterpret_cast<const void*>(resnet128), hipFuncAttributeMaxDynamicSharedMemorySize,shared_memory);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, resnet128, numThreads, shared_memory);
    //hipFuncSetAttribute(reinterpret_cast<const void*>(alexnet32), hipFuncAttributePreferredSharedMemoryCarveout,0);

    void* args[] = {&bconv1_gpu, 
        &l1b1c1_gpu, 
        &l1b1c2_gpu,
        &l1b2c1_gpu,
        &l1b2c2_gpu,
        &l2b1c1_gpu, 
        &l2b1c2_gpu,
        &l2b2c1_gpu,
        &l2b2c2_gpu,
        &l3b1c1_gpu, 
        &l3b1c2_gpu,
        &l3b2c1_gpu,
        &l3b2c2_gpu,
        &bfc1_gpu,
        &bout_gpu};

    START_TIMER;

    hipLaunchCooperativeKernel((void*)resnet128, numBlocksPerSm*deviceProp.multiProcessorCount, 
            numThreads, args, shared_memory);

    STOP_TIMER;


    //================ Output =================
    float* output = bout->download_output();
    //validate_prediction(output, image_labels, output_size, batch);

/*
    float* out = l2b1c2->download_full_output();
    //for (int i=0; i<512; i++)
    for (int i=4096; i<4096+512; i++)
    {
        printf("%.f ", out[i]);
        if ((i+1)%32==0) printf("\n");
    }

    printf("\n===%f===\n", bout->bn_scale[0]);
*/




    delete bconv1;
    delete l1b1c1;
    delete l1b1c2;
    delete l1b2c1;
    delete l1b2c2;

    delete l2b1c1;
    delete l2b1c2;
    delete l2b2c1;
    delete l2b2c2;

    delete l3b1c1;
    delete l3b1c2;
    delete l3b2c1;
    delete l3b2c2;

    delete bfc1;
    delete bout;

    return 0;
}


