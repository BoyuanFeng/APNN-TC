#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// File: benn_scaleup.cu
// A ResNet-18 BNN for ImageNet on multi-GPUs in a single node via NCCL.
// ---------------------------------------------------------------------------
// See our arXiv paper for detail: https://arxiv.org/abs/2006.16578
// Ang Li, Scientist, Pacific Northwest National Laboratory(PNNL), U.S.
// Homepage: http://www.angliphd.com
// GitHub repo: http://www.github.com/pnnl/TCBNN
// PNNL-IPID: 31925-E, ECCN: EAR99, IR: PNNL-SA-152850
// BSD Lincese.
// Richland, 99352, WA, USA. June-30-2020.
// ---------------------------------------------------------------------------

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <iostream>
#include <string>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <nccl.h>
#include <omp.h>
#include "utility.h"
#include "param.h"
#include "kernel.cuh"
#include "data.h"

using namespace cooperative_groups;
using namespace std;


#ifdef NEWFMT

__global__ void resnet128(
        InConv128LayerParam* bconv1, 
        Conv128LayerParam* l1b1c1, 
        Conv128LayerParam* l1b1c2,
        Conv128LayerParam* l1b2c1, 
        Conv128LayerParam* l1b2c2,
        Conv128LayerParam* l2b1c1, 
        Conv128LayerParam* l2b1c2,
        Conv128LayerParam* l2b2c1, 
        Conv128LayerParam* l2b2c2,
        Conv128LayerParam* l3b1c1, 
        Conv128LayerParam* l3b1c2,
        Conv128LayerParam* l3b2c1, 
        Conv128LayerParam* l3b2c2,
        Conv128LayerParam* l4b1c1, 
        Conv128LayerParam* l4b1c2,
        Conv128LayerParam* l4b2c1, 
        Conv128LayerParam* l4b2c2,
        Fc128LayerParam* bfc1, 
        Out128LayerParam* bout)
{
    //SET_KERNEL_TIMER;
    grid_group grid = this_grid();
    //========= Conv1 ============
    InConv128LayerFMT(bconv1);
    grid.sync();
    //TICK_KERNEL_TIMER(bconv1);
    //========= L1B1 ============
    Conv128LayerFMT(l1b1c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l1b1c1);
    Conv128LayerFMT(l1b1c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l1b1c2);
    //========= L1B2 ============
    Conv128LayerFMT(l1b2c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l1b2c1);
    Conv128LayerFMT(l1b2c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l1b2c2);
    //========= L2B1 ============
    Conv128LayerFMT(l2b1c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l2b1c1);
    Conv128LayerFMT(l2b1c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l2b1c2);
    //========= L2B2 ============
    Conv128LayerFMT(l2b2c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l2b2c1);
    Conv128LayerFMT(l2b2c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l2b2c2);
    //========= L3B1 ============
    Conv128LayerFMT(l3b1c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l3b1c1);
    Conv128LayerFMT(l3b1c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l3b1c2);
    //========= L3B2 ============
    Conv128LayerFMT(l3b2c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l3b2c1);
    Conv128LayerFMT(l3b2c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l3b2c2);
    //========= L4B1 ============
    Conv128LayerFMT(l4b1c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l4b1c1);
    Conv128LayerFMT(l4b1c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l4b1c2);
    //========= L4B2 ============
    Conv128LayerFMT(l4b2c1);
    grid.sync();
    //TICK_KERNEL_TIMER(l4b2c1);
    Conv128LayerFMT(l4b2c2);
    grid.sync();
    //TICK_KERNEL_TIMER(l4b2c2);
    //========= Fc1 ============
    Fc128LayerFMT(bfc1);
    grid.sync();
    //TICK_KERNEL_TIMER(bfc1);
    //========== Output ===========
    Out128LayerFMT(bout);
    //grid.sync();
    //TICK_KERNEL_TIMER(bout);
}

#else

__global__ void resnet128(
        InConv128LayerParam* bconv1, 
        Conv128LayerParam* l1b1c1, 
        Conv128LayerParam* l1b1c2,
        Conv128LayerParam* l1b2c1, 
        Conv128LayerParam* l1b2c2,
        Conv128LayerParam* l2b1c1, 
        Conv128LayerParam* l2b1c2,
        Conv128LayerParam* l2b2c1, 
        Conv128LayerParam* l2b2c2,
        Conv128LayerParam* l3b1c1, 
        Conv128LayerParam* l3b1c2,
        Conv128LayerParam* l3b2c1, 
        Conv128LayerParam* l3b2c2,
        Conv128LayerParam* l4b1c1, 
        Conv128LayerParam* l4b1c2,
        Conv128LayerParam* l4b2c1, 
        Conv128LayerParam* l4b2c2,
        Fc128LayerParam* bfc1, 
        Out128LayerParam* bout)
{
    grid_group grid = this_grid();
    //========= Conv1 ============
    InConv128Layer(bconv1);
    grid.sync();
    //========= L1B1 ============
    Conv128Layer(l1b1c1);
    grid.sync();
    Conv128Layer(l1b1c2);
    grid.sync();
    //========= L1B2 ============
    Conv128Layer(l1b2c1);
    grid.sync();
    Conv128Layer(l1b2c2);
    grid.sync();
    //========= L2B1 ============
    Conv128Layer(l2b1c1);
    grid.sync();
    Conv128Layer(l2b1c2);
    grid.sync();
    //========= L2B2 ============
    Conv128Layer(l2b2c1);
    grid.sync();
    Conv128Layer(l2b2c2);
    grid.sync();
    //========= L3B1 ============
    Conv128Layer(l3b1c1);
    grid.sync();
    Conv128Layer(l3b1c2);
    grid.sync();
    //========= L3B2 ============
    Conv128Layer(l3b2c1);
    grid.sync();
    Conv128Layer(l3b2c2);
    grid.sync();
    //========= L4B1 ============
    Conv128Layer(l4b1c1);
    grid.sync();
    Conv128Layer(l4b1c2);
    grid.sync();
    //========= L4B2 ============
    Conv128Layer(l4b2c1);
    grid.sync();
    Conv128Layer(l4b2c2);
    grid.sync();
    //========= Fc1 ============
    Fc128Layer(bfc1);
    grid.sync();
    //========== Output ===========
    Out128Layer(bout);
}

#endif

void run(unsigned batch, float* pcomp_time, float* pcomm_time)
{
    const int n_gpu = 2;
    float comp_times[8] = {0};
    float comm_times[8] = {0};
    //printf("EBNN with ResNet-18 on ImageNet for Scaling-Up using %d GPUs!\n", n_gpu);

    ncclUniqueId id;
    ncclGetUniqueId(&id);
    ncclComm_t comm[n_gpu];

    //const unsigned batch = 64;
    const unsigned output_size = 1000;
    const unsigned image_height = 224;
    const unsigned image_width = 224;
    const unsigned image_channel = 3;

    //=============== Get Input and Label =================
    float* images = (float*)malloc(batch*image_height*image_width*image_channel*sizeof(float));
    unsigned* image_labels = (unsigned*)malloc(batch*sizeof(unsigned));
    read_ImageNet_normalized("./imagenet_files.txt", images, image_labels, batch);




//Start OpenMP
#pragma omp parallel num_threads (n_gpu) shared (comm, id, batch, images, image_labels)
{
    int i_gpu = omp_get_thread_num();
    CUDA_SAFE_CALL(hipSetDevice(i_gpu));
    hipEvent_t comp_start, comp_stop, comm_start, comm_stop;
    CUDA_SAFE_CALL(hipEventCreate(&comp_start));
    CUDA_SAFE_CALL(hipEventCreate(&comp_stop));
    CUDA_SAFE_CALL(hipEventCreate(&comm_start));
    CUDA_SAFE_CALL(hipEventCreate(&comm_stop));
    CHECK_NCCL(ncclCommInitRank(&comm[i_gpu], n_gpu, id, i_gpu));

    int dev = i_gpu;
    //================ Get Weight =================
    FILE* config_file = fopen("./resnet_imagenet.csv","r");

    //================ Set Network =================
    //Layer-0
    InConv128LayerParam* bconv1 = new InConv128LayerParam("Conv1", image_height, image_width, 
            7, 7, 3, 64, batch,4,4,true,1,1,true);//save residual 
    InConv128LayerParam* bconv1_gpu = bconv1->initialize(images, config_file);

    //Layer-1, basic-block-1, conv1
    Conv128LayerParam* l1b1c1 = new Conv128LayerParam("L1B1C1", bconv1->output_height, 
            bconv1->output_width, 3, 3, 64, 64, batch);
    Conv128LayerParam* l1b1c1_gpu = l1b1c1->initialize(config_file, bconv1->get_output_gpu());

    //Layer-1, basic-block-1, conv2
    Conv128LayerParam* l1b1c2 = new Conv128LayerParam("L1B1C2", l1b1c1->output_height, 
            l1b1c1->output_width, 3, 3, 64, 64, batch,1,1,true,1,1,false,true,true,64);
    Conv128LayerParam* l1b1c2_gpu = l1b1c2->initialize(config_file, l1b1c1->get_output_gpu(),
            bconv1->get_output_residual_gpu());

    //Layer-1, basic-block-2, conv1
    Conv128LayerParam* l1b2c1 = new Conv128LayerParam("L1B2C1", l1b1c2->output_height, 
            l1b1c2->output_width, 3, 3, 64, 64, batch);
    Conv128LayerParam* l1b2c1_gpu = l1b2c1->initialize(config_file, l1b1c2->get_output_gpu());

    //Layer-1, basic-block-2, conv2
    Conv128LayerParam* l1b2c2 = new Conv128LayerParam("L1B2C2", l1b2c1->output_height, 
            l1b2c1->output_width, 3, 3, 64, 64, batch,1,1,true,1,1,false,true,true,128);
    Conv128LayerParam* l1b2c2_gpu = l1b2c2->initialize(config_file, l1b2c1->get_output_gpu(),
            l1b1c2->get_output_residual_gpu());

    //=============
    //Layer-2, basic-block-1, conv1
    Conv128LayerParam* l2b1c1 = new Conv128LayerParam("L2B1C1", l1b2c2->output_height, 
            l1b2c2->output_width, 3, 3, 64, 128, batch, 2, 2);
    Conv128LayerParam* l2b1c1_gpu = l2b1c1->initialize(config_file, l1b2c2->get_output_gpu());

    //Layer-2, basic-block-1, conv2
    Conv128LayerParam* l2b1c2 = new Conv128LayerParam("L2B1C2", l2b1c1->output_height, 
            l2b1c1->output_width, 3, 3, 128, 128, batch,1,1,true,1,1,false,true,true,128,true);
    Conv128LayerParam* l2b1c2_gpu = l2b1c2->initialize(config_file, l2b1c1->get_output_gpu(),
            l1b2c2->get_output_residual_gpu());

    //Layer-2, basic-block-2, conv1
    Conv128LayerParam* l2b2c1 = new Conv128LayerParam("L2B2C1", l2b1c2->output_height, 
            l2b1c2->output_width, 3, 3, 128, 128, batch, 1, 1);
    Conv128LayerParam* l2b2c1_gpu = l2b2c1->initialize(config_file, l2b1c2->get_output_gpu());

    //Layer-2, basic-block-2, conv2
    Conv128LayerParam* l2b2c2 = new Conv128LayerParam("L2B2C2", l2b2c1->output_height, 
            l2b2c1->output_width, 3, 3, 128, 128, batch,1,1,true,1,1,false,true,true,128);
    Conv128LayerParam* l2b2c2_gpu = l2b2c2->initialize(config_file, l2b2c1->get_output_gpu(),
            l2b1c2->get_output_residual_gpu());

    //=============
    //Layer-3, basic-block-1, conv1
    Conv128LayerParam* l3b1c1 = new Conv128LayerParam("L3B1C1", l2b2c2->output_height, 
            l2b2c2->output_width, 3, 3, 128, 256, batch, 2, 2);
    Conv128LayerParam* l3b1c1_gpu = l3b1c1->initialize(config_file, l2b2c2->get_output_gpu());

    //Layer-3, basic-block-1, conv2
    Conv128LayerParam* l3b1c2 = new Conv128LayerParam("L3B1C2", l3b1c1->output_height, 
            l3b1c1->output_width, 3, 3, 256, 256, batch,1,1,true,1,1,false,true,true,128,true);
    Conv128LayerParam* l3b1c2_gpu = l3b1c2->initialize(config_file, l3b1c1->get_output_gpu(),
            l2b2c2->get_output_residual_gpu());

    //Layer-3, basic-block-2, conv1
    Conv128LayerParam* l3b2c1 = new Conv128LayerParam("L3B2C1", l3b1c2->output_height, 
            l3b1c2->output_width, 3, 3, 256, 256, batch, 1, 1);
    Conv128LayerParam* l3b2c1_gpu = l3b2c1->initialize(config_file, l3b1c2->get_output_gpu());

    //Layer-3, basic-block-2, conv2
    Conv128LayerParam* l3b2c2 = new Conv128LayerParam("L3B2C2", l3b2c1->output_height, 
            l3b2c1->output_width, 3, 3, 256, 256, batch,1,1,true,1,1,false,true,true,256);
    Conv128LayerParam* l3b2c2_gpu = l3b2c2->initialize(config_file, l3b2c1->get_output_gpu(),
            l3b1c2->get_output_residual_gpu());

    //=============
    //Layer-4, basic-block-1, conv1
    Conv128LayerParam* l4b1c1 = new Conv128LayerParam("L4B1C1", l3b2c2->output_height, 
            l3b2c2->output_width, 3, 3, 256, 512, batch, 2, 2);
    Conv128LayerParam* l4b1c1_gpu = l4b1c1->initialize(config_file, l3b2c2->get_output_gpu());

    //Layer-4, basic-block-1, conv2
    Conv128LayerParam* l4b1c2 = new Conv128LayerParam("L4B1C2", l4b1c1->output_height, 
            l4b1c1->output_width, 3, 3, 512, 512, batch,1,1,true,1,1,false,true,true,256,true);
    Conv128LayerParam* l4b1c2_gpu = l4b1c2->initialize(config_file, l4b1c1->get_output_gpu(),
            l3b2c2->get_output_residual_gpu());

    //Layer-4, basic-block-2, conv1
    Conv128LayerParam* l4b2c1 = new Conv128LayerParam("L4B2C1", l4b1c2->output_height, 
            l4b1c2->output_width, 3, 3, 512, 512, batch, 1, 1);
    Conv128LayerParam* l4b2c1_gpu = l4b2c1->initialize(config_file, l4b1c2->get_output_gpu());

    //Layer-4, basic-block-2, conv2
    Conv128LayerParam* l4b2c2 = new Conv128LayerParam("L4B2C2", l4b2c1->output_height, 
            l4b2c1->output_width, 3, 3, 512, 512, batch,1,1,true,1,1,true,false,true,512);
    Conv128LayerParam* l4b2c2_gpu = l4b2c2->initialize(config_file, l4b2c1->get_output_gpu(),
            l4b1c2->get_output_residual_gpu());

    //=============
    //Layer-5
    Fc128LayerParam* bfc1 = new Fc128LayerParam("Fc1", batch, (l4b2c2->output_height)
            *(l4b2c2->output_width)*512, 512); 
    Fc128LayerParam* bfc1_gpu = bfc1->initialize(config_file, l4b2c2->get_output_gpu());
    //Out Layer
    Out128LayerParam* bout = new Out128LayerParam("Fout", batch, 512, output_size);
    Out128LayerParam* bout_gpu = bout->initialize(config_file, bfc1->get_output_gpu());  

    //================ Setup Kernel =================
    int numThreads = 1024;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int numBlocksPerSm;
    int shared_memory = 512*sizeof(int)*32;
    hipFuncSetAttribute(reinterpret_cast<const void*>(resnet128), hipFuncAttributeMaxDynamicSharedMemorySize,shared_memory);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, resnet128, numThreads, shared_memory);

    //hipFuncSetAttribute(reinterpret_cast<const void*>(resnet128), hipFuncAttributePreferredSharedMemoryCarveout,0);

    void* args[] = {&bconv1_gpu, 
        &l1b1c1_gpu, 
        &l1b1c2_gpu,
        &l1b2c1_gpu,
        &l1b2c2_gpu,
        &l2b1c1_gpu, 
        &l2b1c2_gpu,
        &l2b2c1_gpu,
        &l2b2c2_gpu,
        &l3b1c1_gpu, 
        &l3b1c2_gpu,
        &l3b2c1_gpu,
        &l3b2c2_gpu,
        &l4b1c1_gpu, 
        &l4b1c2_gpu,
        &l4b2c1_gpu,
        &l4b2c2_gpu,
        &bfc1_gpu,
        &bout_gpu};


    CUDA_SAFE_CALL(hipDeviceSynchronize());
    #pragma omp barrier
    //START_TIMER;
    
    hipEventRecord(comp_start);

    hipLaunchCooperativeKernel((void*)resnet128, numBlocksPerSm*deviceProp.multiProcessorCount, 
            numThreads, args, shared_memory);
    
    hipEventRecord(comp_stop);

    //if (i_gpu == 0) hipMemset(bout->get_output_gpu(), 0, bout->output_bytes());
    //if (i_gpu == 1) hipMemset(bout->get_output_gpu(), 0, bout->output_bytes());

    hipEventRecord(comm_start);

    CHECK_NCCL( ncclReduce(bout->get_output_gpu(), bout->get_output_gpu(), 
                bout->output_size(), ncclFloat, ncclMax, 0, comm[i_gpu], 0) );

    hipEventRecord(comm_stop);

    //STOP_TIMER;
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    float comp_time, comm_time;
    
    hipEventElapsedTime(&comp_time,comp_start,comp_stop);
    hipEventElapsedTime(&comm_time,comm_start,comm_stop);

    comp_times[i_gpu] =  comp_time;
    comm_times[i_gpu] =  comm_time;

    ncclCommDestroy(comm[i_gpu]);
    //CHECK_NCCL(ncclCommDestroy(comm[i_gpu]));

    //================ Output =================
    //if (i_gpu == 0)
    //{
    //float* output = bout->download_output();
    //validate_prediction(output, image_labels, output_size, batch);
    //}


/*
    float* out = l1b2c1->download_full_output();
    //float* out = l1b1c2->download_full_output();
    //for (int i=0; i<512; i++)
    for (int i=4096; i<4096+512; i++)
    {
        printf("%.f ", out[i]);
        if ((i+1)%32==0) printf("\n");
    }
    printf("\n===%f===\n", bout->bn_scale[0]);
*/ 

    delete bconv1;
    delete l1b1c1;
    delete l1b1c2;
    delete l1b2c1;
    delete l1b2c2;

    delete l2b1c1;
    delete l2b1c2;
    delete l2b2c1;
    delete l2b2c2;

    delete l3b1c1;
    delete l3b1c2;
    delete l3b2c1;
    delete l3b2c2;

    delete l4b1c1;
    delete l4b1c2;
    delete l4b2c1;
    delete l4b2c2;

    delete bfc1;
    delete bout;
} //end of omp

    float max_comp_time = 0;
    float max_comm_time = 0;
    for (int i=0; i<n_gpu; i++)
    {
        if (comp_times[i]>max_comp_time)
            max_comp_time = comp_times[i];
        if (comm_times[i]>max_comm_time)
            max_comm_time = comm_times[i];
    }
    *pcomp_time = max_comp_time;
    *pcomm_time = max_comm_time;
}



int main()
{
    const int tests = 2;
    for (int b=256; b<512; b*=2)
    {
        float avg_comp_time = 0;
        float avg_comm_time = 0;
        float comp_time, comm_time;
        for (int i=0; i<tests; i++)
        {
            run(b,&comp_time,&comm_time);
            avg_comp_time += comp_time;
            avg_comm_time += comm_time;
        }
        avg_comp_time /= float(tests);
        avg_comm_time /= float(tests);
        printf("Scale_up_test: Batch=%d, Comp_Time=%.3f, Comm_Time:%.3f \n",
                b,avg_comp_time, avg_comm_time);
    }
    return 0;
}




















