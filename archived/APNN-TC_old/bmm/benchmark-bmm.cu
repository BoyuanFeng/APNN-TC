#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  benchmark-bmm.cu
 *
 *    Description:  Accelerate BNN via TensorCores in Turing/Ampere GPU
 *                  Please see our TPDS paper "Accelerating Binarized Neural 
 *                  Networks via Bit-Tensor-Cores in Turing GPUs" for detail.
 *                  https://arxiv.org/abs/2006.16578
 *
 *        Version:  1.0
 *        Created:  11/04/2019 11:43:58 AM, Richland, WA, USA.
 *       Compiler:  nvcc -arch=sm_75
 *       
 *      PNNL-IPID: 31925-E, ECCN: EAR99, IR: PNNL-SA-152850.
 *
 *         Author:  Ang Li
 *        Website:  https://www.angliphd.com
 *
 * =====================================================================================
 */


#include <iostream>
#include <hipblas.h>
#include <sys/time.h>

#include "binary_kernels.cu"
#include "matmul_kernel.cu"
#include "tensorcore_kernel.cu"

#define TEST_TIMES 10

using namespace std;

bool check_result(float* p1, float* p2, const int N) 
{
    bool flag = true;
    for (int i = 0; i < N * N; i ++) {
        //printf("(%f,%f)",p1[i],p2[i]);
        float diff = p1[i] - p2[i];
        if (fabs(diff) > 1e-6) {
            flag = false;
        }
    }
    return flag;
}

bool check_result(float* p1, int* p2, const int N) 
{
    bool flag = true;
    for (int i = 0; i < N * N; i ++) {
        //printf("(%.0f,%d)",p1[i],p2[i]);
        float diff = p1[i] - (float)p2[i];
        if (fabs(diff) > 1e-6) {
            flag = false;
        }
    }
    return flag;
}



int main(int argc, char* argv[]) 
{
    bool trans_A = false;
    bool trans_B = false;

    hipSetDevice(0);

    if (argc != 2) 
    {
        printf("./exe N\n");
        exit(1);
    }
    int N = atoi(argv[1]);
    srand(time(0));
	// prepare data
	float *A = (float*)malloc(N * N * sizeof(float));
	float *B = (float*)malloc(N * N * sizeof(float));
	for (int i = 0; i < N * N; i ++) 
    {
        double x = (double)rand() / RAND_MAX;
        A[i] = (x > 0.5) ? 1 : -1;
        x = (double)rand() / RAND_MAX;
        B[i] = (x > 0.5) ? 1 : -1;
    }

	// copy to cuda
	float *fA, *fB, *fC;
    unsigned *uC;
    ullong *ullC;
	hipMalloc(&fA, N * N * sizeof(float));
	hipMalloc(&fB, N * N * sizeof(float));
	hipMalloc(&fC, N * N * sizeof(float));
	hipMalloc(&uC, N * N * sizeof(unsigned));
	hipMalloc(&ullC, N * N * sizeof(unsigned long long));
	hipMemcpy(fA, A, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(fB, B, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    //============================================= XNOR
    unsigned int *Aconc, *Bconc;
    hipMalloc(&Aconc, N * N);
    hipMalloc(&Bconc, N * N);
    hipMemset(fC, 0, N * N * sizeof(float));
    int block = 64, grid = N * N / (block * 32)  + 1;
    int grid1 = N / block + 1;
    dim3 blockDim(16, 16);
    dim3 gridDim(N / 16 + 1, N / 16 + 1);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        concatenate_rows_kernel<<<grid, block>>>(fA, Aconc, N * N / 32);
        concatenate_cols_kernel<<<grid1, block>>>(fB, Bconc, N, N);
        xnor_gemm<<<gridDim, blockDim>>>(Aconc, Bconc, fC, N, N / 32, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double baseline_time = (milliseconds*1e3)/double(TEST_TIMES);
    //----------------------- 

    hipFree(Aconc);
    hipFree(Bconc);
    float* result_xnor = (float*)malloc(N * N * sizeof(float));
    hipMemcpy(result_xnor, fC, N * N * sizeof(float), hipMemcpyDeviceToHost);

    //============================================= CUBLAS
    hipMemset(fC, 0, N * N * sizeof(int));
    hipblasOperation_t cublas_trans_A = trans_A?HIPBLAS_OP_T:HIPBLAS_OP_N;
    hipblasOperation_t cublas_trans_B = trans_B?HIPBLAS_OP_T:HIPBLAS_OP_N;
    __half* hfA = NULL; 
    hipMalloc(&hfA, N*N*sizeof(__half));
    hipMemset(hfA, 0, N*N*sizeof(__half));
    __half* hfB = NULL;
    hipMalloc(&hfB, N*N*sizeof(__half));
    hipMemset(hfB, 0, N*N*sizeof(__half));
    __half* hfC = NULL;
    hipMalloc(&hfC, N*N*sizeof(__half));
    hipMemset(hfC, 0, N*N*sizeof(__half));


    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0, beta = 0.0;
    __half hf_alpha = __float2half(alpha);
    __half hf_beta = __float2half(beta);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // cublas use column-major
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        float_to_half<<<N*N/256,256>>>(fA, hfA, N*N);
        float_to_half<<<N*N/256,256>>>(fB, hfB, N*N);
        //hipblasSgemm(handle, cublas_trans_A, cublas_trans_B, N, N, N,
        //&alpha, fB, N, fA, N, &beta, fC, N);
        hipblasHgemm(handle, cublas_trans_A, cublas_trans_B, N, N, N,
                &hf_alpha, hfB, N, hfA, N, &hf_beta, hfC, N);
        half_to_float<<<N*N/256,256>>>(hfC, fC, N*N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);


    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double cublas_time = (milliseconds*1e3)/double(TEST_TIMES);
    //----------------------- 

    float* result_cublas = (float*)malloc(N * N * sizeof(float));
    hipMemcpy(result_cublas, fC, N * N * sizeof(float), hipMemcpyDeviceToHost);
    
    
    //============================================= BSTC-32
    hipMemset(fC, 0, N * N * sizeof(float));
    
    unsigned *tA, *tB;
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));
    dim3 bmmDim(N/32, N/32);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        ToBit32Row<float><<<dim3(N/32,N/32), 32>>>(fB, tB, N, N);
        ToBit32Col<float><<<dim3(N/32,N/32), 32>>>(fA, tA, N, N);
        BMM32_Arow_Brow<float><<<dim3(N/32,N/32), 32>>>(tA, tB, fC, N, N/32, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bblas_time = (milliseconds*1e3)/double(TEST_TIMES);

    hipFree(tA);
    hipFree(tB);
    //----------------------- 

    float* result_bblas = (float*)malloc(N * N * sizeof(float));
    hipMemcpy(result_bblas, fC, N * N * sizeof(float), hipMemcpyDeviceToHost);

    //============================================= BBLAS-64
    hipMemset(fC, 0, N * N * sizeof(float));

    ullong *llA, *llB;
	hipMalloc(&llA, N * N/64 * sizeof(ullong));
	hipMalloc(&llB, N * N/64 * sizeof(ullong));

    dim3 bmm64Dim(N/64, N/64);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        ToBit64Row<float><<<dim3(N/64,N/32), 32>>>(fB, llB, N, N);
        ToBit64Col<float><<<dim3(N/32,N/64), 32>>>(fA, llA, N, N);
        BMM64_Arow_Brow<float><<<dim3(N/64,N/64), 32>>>(llA, llB, fC, N, N/64, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double b64blas_time = (milliseconds*1e3)/double(TEST_TIMES);
    //----------------------- 
    float* result_b64blas = (float*)malloc(N * N * sizeof(float));
    hipMemcpy(result_b64blas, fC, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(llA);
    hipFree(llB);

    //============================================= BSTC-32-Small
    hipMemset(fC, 0, N * N * sizeof(float));
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        ToBit32Row<float><<<dim3(N/32,N/32), 32>>>(fB, tB, N, N);
        ToBit32Col<float><<<dim3(N/32,N/32), 32>>>(fA, tA, N, N);
        BMM32_MT_M_S<<<dim3(N/32,N/32), 1024>>>(tA, tB, fC, N, N/32, N);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmm32s_time = (milliseconds*1e3)/double(TEST_TIMES);
    hipFree(tA);
    hipFree(tB);
    //----------------------- 
    float* result_bmm32s = (float*)malloc(N * N * sizeof(float));
    hipMemcpy(result_bmm32s, fC, N * N * sizeof(float), hipMemcpyDeviceToHost);

    //============================================= BSTC-64-Small
    hipMemset(fC, 0, N * N * sizeof(float));
	hipMalloc(&llA, N * N/64 * sizeof(ullong));
	hipMalloc(&llB, N * N/64 * sizeof(ullong));
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        ToBit64Row<float><<<dim3(N/64,N/32), 32>>>(fB, llB, N, N);
        ToBit64Col<float><<<dim3(N/32,N/64), 32>>>(fA, llA, N, N);
        BMM64_MT_M_S<<<dim3(N/64,N/64), 1024>>>(llA, llB, fC, N, N/64, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmm64s_time = (milliseconds*1e3)/double(TEST_TIMES);
    //----------------------- 
    float* result_bmm64s = (float*)malloc(N * N * sizeof(float));
    hipMemcpy(result_bmm64s, fC, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(llA);
    hipFree(llB);

    //============================================= BSTC-32-Bin
    hipMemset(uC, 0, N * N * sizeof(unsigned));
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));
    ToBit32Row<float><<<dim3(N/32,N/32), 32>>>(fB, tB, N, N);
    ToBit32Col<float><<<dim3(N/32,N/32), 32>>>(fA, tA, N, N);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMM32_BIN<<<dim3(N/32,N/32), 32>>>(tA, tB, uC, N, N, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmm32_bin_time = (milliseconds*1e3)/double(TEST_TIMES);
    hipFree(tA);
    hipFree(tB);
    //----------------------- 
    unsigned* result_bmm32_bin = (unsigned*)malloc(N * N * sizeof(unsigned));
    hipMemcpy(result_bmm32_bin, uC, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);

    //============================================= BSTC-64-Bin
    hipMemset(ullC, 0, N * N * sizeof(ullong));
	hipMalloc(&llA, N * N/64 * sizeof(ullong));
	hipMalloc(&llB, N * N/64 * sizeof(ullong));
    ToBit64Row<float><<<dim3(N/64,N/32), 32>>>(fB, llB, N, N);
    ToBit64Col<float><<<dim3(N/32,N/64), 32>>>(fA, llA, N, N);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMM64_BIN<<<dim3(N/64,N/64), 32>>>(llA, llB, ullC, N, N, N);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmm64_bin_time = (milliseconds*1e3)/double(TEST_TIMES);
    //----------------------- 
    ullong* result_bmm64_bin = (ullong*)malloc(N * N * sizeof(ullong));
    hipMemcpy(result_bmm64_bin, ullC, N * N * sizeof(ullong), hipMemcpyDeviceToHost);
    hipFree(llA);
    hipFree(llB);

    //============================================= BSTC-32-Small-Bin
    hipMemset(uC, 0, N * N * sizeof(unsigned));
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));
    ToBit32Row<float><<<dim3(N/32,N/32), 32>>>(fB, tB, N, N);
    ToBit32Col<float><<<dim3(N/32,N/32), 32>>>(fA, tA, N, N);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMM32S_BIN<<<dim3(N/32,N/32), 1024>>>(tA, tB, uC, N, N, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmm32s_bin_time = (milliseconds*1e3)/double(TEST_TIMES);
    hipFree(tA);
    hipFree(tB);
    //----------------------- 
    unsigned* result_bmm32s_bin = (unsigned*)malloc(N * N * sizeof(unsigned));
    hipMemcpy(result_bmm32s_bin, uC, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);



    //============================================= BSTC-64-Small-Bin
    hipMemset(ullC, 0, N * N * sizeof(ullong));
	hipMalloc(&llA, N * N/64 * sizeof(ullong));
	hipMalloc(&llB, N * N/64 * sizeof(ullong));
    ToBit64Row<float><<<dim3(N/64,N/32), 32>>>(fB, llB, N, N);
    ToBit64Col<float><<<dim3(N/32,N/64), 32>>>(fA, llA, N, N);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMM64S_BIN<<<dim3(N/64,N/64), 1024>>>(llA, llB, ullC, N, N/64, N);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmm64s_bin_time = (milliseconds*1e3)/double(TEST_TIMES);
    //----------------------- 
    ullong* result_bmm64s_bin = (ullong*)malloc(N * N * sizeof(ullong));
    hipMemcpy(result_bmm64s_bin, ullC, N * N * sizeof(ullong), hipMemcpyDeviceToHost);
    hipFree(llA);
    hipFree(llB);

    //============================================= TensorCore
    int* tC = NULL;
    hipMalloc(&tC, N * N * sizeof(int));
    hipMemset(tC, 0, N * N * sizeof(int));
    
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));
    const unsigned BMMA_M = 4;
    const unsigned BMMA_N = 4;
    dim3 tensorcoreBlk(32, BMMA_M, BMMA_N);
    dim3 tensorcoreDim(N/(8*BMMA_M), N/(8*BMMA_N));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMMA_toBit32Row<float><<<dim3(N,N/32), 32>>>(fA, tA, N, N);
        BMMA_toBit32Col<float><<<dim3(N/32,N), 32>>>(fB, tB, N, N);
        BMMApipe<BMMA_M,BMMA_N><<<tensorcoreDim, tensorcoreBlk>>>(tA, tB, tC, N, N, N/128);
        //BMMA<BMMA_M,BMMA_N><<<tensorcoreDim, tensorcoreBlk>>>(tA, tB, tC, N, N, N/128);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double tensorcore_time = (milliseconds*1e3)/double(TEST_TIMES);

    hipFree(tA);
    hipFree(tB);
    //----------------------- 

    int* result_tensorcore = (int*)malloc(N * N * sizeof(int));
    memset(result_tensorcore, 0, N*N*sizeof(int));
    hipMemcpy(result_tensorcore, tC, N * N * sizeof(int), hipMemcpyDeviceToHost);


    //============================================= TensorCore_Small
    hipMemset(tC, 0, N * N * sizeof(int));
    
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));
    const unsigned BMMAS_M = 4;

    dim3 tensorcoreSBlk(32, 2);
    dim3 tensorcoreSDim(N/16, N/8);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMMA_toBit32Row<float><<<dim3(N,N/32), 32>>>(fA, tA, N, N);
        BMMA_toBit32Col<float><<<dim3(N/32,N), 32>>>(fB, tB, N, N);
        BMMAS<BMMAS_M><<<tensorcoreSDim, tensorcoreSBlk>>>(tA, tB, tC, N, N, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double tensorcore_s_time = (milliseconds*1e3)/double(TEST_TIMES);

    hipFree(tA);
    hipFree(tB);
    //----------------------- 

    int* result_tensorcore_s = (int*)malloc(N * N * sizeof(int));
    hipMemcpy(result_tensorcore_s, tC, N * N * sizeof(int), hipMemcpyDeviceToHost);


    //============================================= TensorCore_Bin
    hipMemset(uC, 0, N * N * sizeof(unsigned));
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));

    //const unsigned BMMA_M = 4;
    //const unsigned BMMA_N = 4;
    //dim3 tensorcoreBlk(32, BMMA_M, BMMA_N);
    //dim3 tensorcoreDim(N/(8*BMMA_M), N/(8*BMMA_N));
    BMMA_toBit32Row<float><<<dim3(N,N/32), 32>>>(fA, tA, N, N);
    BMMA_toBit32Col<float><<<dim3(N/32,N), 32>>>(fB, tB, N, N);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMMA_bin<BMMA_M,BMMA_N><<<tensorcoreDim, tensorcoreBlk>>>(tA, tB, uC, N, N, N/128);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmma_bin_time = (milliseconds*1e3)/double(TEST_TIMES);
    hipFree(tA);
    hipFree(tB);
    //----------------------- 
    unsigned* result_bmma_bin = (unsigned*)malloc(N * N * sizeof(unsigned));
    hipMemcpy(result_bmma_bin, uC, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);



    //============================================= TensorCore_Small_Bin
    hipMemset(uC, 0, N * N * sizeof(unsigned));
    
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));
    
    BMMA_toBit32Row<float><<<dim3(N,N/32), 32>>>(fA, tA, N, N);
    BMMA_toBit32Col<float><<<dim3(N/32,N), 32>>>(fB, tB, N, N);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMMAS_bin<BMMAS_M><<<tensorcoreSDim, tensorcoreSBlk>>>(tA, tB, uC, N, N, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmmas_bin_time = (milliseconds*1e3)/double(TEST_TIMES);

    hipFree(tA);
    hipFree(tB);
    //----------------------- 

    int* result_bmmas_bin = (int*)malloc(N * N * sizeof(int));
    hipMemcpy(result_bmmas_bin, uC, N * N * sizeof(int), hipMemcpyDeviceToHost);


    //============================================= TensorCore_Small_New_format
    hipMemset(tC, 0, N * N * sizeof(int));
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));

    dim3 tensorcoreSNBlk(32, 2);
    dim3 tensorcoreSNDim(N/16, N/8);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMMA_toBit32Row_new<float><<<dim3(N/8,N/128), dim3(32,8,4)>>>(fA, tA, N, N);
        BMMA_toBit32Col_new<float><<<dim3(N/128,N/8), dim3(32,4,8)>>>(fB, tB, N, N);
        BMMAS_new<<<tensorcoreSNDim, tensorcoreSNBlk>>>(tA, tB, tC, N, N, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmma_sn_time = (milliseconds*1e3)/double(TEST_TIMES);

    hipFree(tA);
    hipFree(tB);
    //----------------------- 

    int* result_bmma_sn = (int*)malloc(N * N * sizeof(int));
    hipMemcpy(result_bmma_sn, tC, N * N * sizeof(int), hipMemcpyDeviceToHost);


    //============================================= TensorCore_Small_Bin_New_Format
    hipMemset(uC, 0, N * N * sizeof(unsigned));
    
	hipMalloc(&tA, N * N/32 * sizeof(unsigned));
	hipMalloc(&tB, N * N/32 * sizeof(unsigned));
    
    BMMA_toBit32Row_new<float><<<dim3(N/8,N/128), dim3(32,8,4)>>>(fA, tA, N, N);
    BMMA_toBit32Col_new<float><<<dim3(N/128,N/8), dim3(32,4,8)>>>(fB, tB, N, N);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    //----------------------- 
    hipEventRecord(start);
    for (int i=0; i<TEST_TIMES; i++)
    {
        BMMAS_bin_new<<<tensorcoreSDim, tensorcoreSBlk>>>(tA, tB, uC, N, N, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    double bmmasn_bin_time = (milliseconds*1e3)/double(TEST_TIMES);

    hipFree(tA);
    hipFree(tB);
    //----------------------- 

    int* result_bmmasn_bin = (int*)malloc(N * N * sizeof(int));
    hipMemcpy(result_bmmasn_bin, uC, N * N * sizeof(int), hipMemcpyDeviceToHost);

    //============================================= CHECK RESULT
    printf("XNOR success: %d\n", check_result(result_cublas, result_xnor, N));
    printf("CUBLAS success: %d\n", check_result(result_cublas, result_cublas, N));
    printf("BMM-32 success: %d\n", check_result(result_cublas, result_bblas, N));
    printf("BMM-64 success: %d\n", check_result(result_cublas, result_b64blas, N));
    printf("BMMS-32 success: %d\n", check_result(result_cublas, result_bmm32s, N));
    printf("BMMS-64 success: %d\n", check_result(result_cublas, result_bmm64s, N));
    printf("BMMA success: %d\n", check_result(result_cublas, result_tensorcore, N));
    printf("BMMAS success: %d\n", check_result(result_cublas, result_tensorcore_s, N));


    //for (int i=0; i<10; i++)
    //printf("bmm32_bin:%x,bmm32s_bin:%x,bmm64_bin:%llx,bmm64s_bin:%llx\n", result_bmm32_bin[i],
    //result_bmm32s_bin[i], result_bmm64_bin[i], result_bmm64s_bin[i]);

    printf("CUBLAS:%.3lf, BNN:%.3lf, BMM-32:%.3lf, BMM-64:%.3lf, BMMS-32:%.3lf, BMMS-64:%.3lf, BMM-32-Bin:%.3lf, BMM-64-Bin:%.3lf, BMMS-32-Bin:%.3lf, BMMS-64-Bin:%.3lf, BMMA:%.3lf, BMMAS:%.3lf, BMMA-Bin:%.3lf, BMMAS-Bin:%.3lf, BMMASN:%.3lf, BMMASN-Bin:%.3lf \n", 
            cublas_time, baseline_time, bblas_time, b64blas_time, bmm32s_time, bmm64s_time,
            bmm32_bin_time, bmm64_bin_time, bmm32s_bin_time, bmm64s_bin_time, tensorcore_time,
            tensorcore_s_time, bmma_bin_time, bmmas_bin_time, bmma_sn_time, bmmasn_bin_time);


    hipFree(fA);
    hipFree(fB);
    hipFree(fC);
    hipFree(uC);
    hipFree(ullC);
    free(result_xnor);
    free(result_cublas);
    free(result_bblas);
    free(result_b64blas);

    free(result_tensorcore);
    free(result_tensorcore_s);
    free(result_bmma_bin);
    free(result_bmmas_bin);
    free(result_bmma_sn);
    free(result_bmmasn_bin);

}
