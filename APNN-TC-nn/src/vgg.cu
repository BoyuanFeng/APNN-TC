#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <iostream>
#include <string>
#include <iostream>
#include <fstream>
#include <vector>

#include "utility.h"
#include "param.h"
#include "kernel.cuh"
#include "data.h"

using namespace std;
     
int main()
{
    int dev = 0;
    hipSetDevice(dev);

    const unsigned batch = 8;
    const unsigned output_size = 1000;
    const unsigned image_height = 224;
    const unsigned image_width = 224;
    const unsigned image_channel = 3;
    const unsigned filter_height = 3;
    const unsigned filter_width = 3;
    const unsigned n_hidden = 4096;

    //=============== Get Input and Label =================
    float* images = (float*)malloc(batch*image_height*image_width*image_channel*sizeof(float));
    unsigned* image_labels = (unsigned*)malloc(batch*sizeof(unsigned));
//     read_ImageNet_normalized("./imagenet_files.txt", images, image_labels, batch);
    
    //================ Get Weight =================
    FILE* config_file = fopen("./vgg_imagenet.csv","r");
    //================ Set Network =================
    //Bconv1 Layer
    // InConv128LayerParam* bconv1 = new InConv128LayerParam("Conv1", image_height, image_width, 
    //         filter_height, filter_width, 3, 64, batch); 
    // InConv128LayerParam* bconv1_gpu = bconv1->initialize(images, config_file);

    uin32* lowBit_image_gpu = images_quantization(images, batch, image_height, image_width, image_channel);
    
    
    Conv128LayerParam* bconv1 = new Conv128LayerParam("Conv1", image_height, image_width, 
        filter_height, filter_width, 3, 96, batch, 2, 2, true, 2, 2); 
    Conv128LayerParam* bconv1_gpu = bconv1->initialize(config_file, lowBit_image_gpu);



    //Bconv2 Layer
    Conv128LayerParam* bconv2 = new Conv128LayerParam("Conv2", bconv1->output_height, 
            bconv1->output_width, filter_height, filter_width, 96, 256, batch, 1, 1,
            true, 1, 1, false, 
            false, false, 0, false, a_bit, w_bit
        );    
    Conv128LayerParam* bconv2_gpu = bconv2->initialize(config_file, bconv1->get_output_gpu());
    //Bconv3 Layer
    Conv128LayerParam* bconv3 = new Conv128LayerParam("Conv3", bconv2->output_height, 
            bconv2->output_width, filter_height, filter_width, 256, 256, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv3_gpu = bconv3->initialize(config_file, bconv2->get_output_gpu());
    //Bconv4 Layer
    Conv128LayerParam* bconv4 = new Conv128LayerParam("Conv4", bconv3->output_height, 
            bconv3->output_width, filter_height, filter_width, 256, 256, batch, 1, 1,
            true, 2, 2, false,
            false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv4_gpu = bconv4->initialize(config_file, bconv3->get_output_gpu());
    
    
    //Bconv5 Layer
    Conv128LayerParam* bconv5 = new Conv128LayerParam("Conv5", bconv4->output_height, 
            bconv4->output_width, filter_height, filter_width, 256, 512, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv5_gpu = bconv5->initialize(config_file, bconv4->get_output_gpu());
    //Bconv6 Layer
    Conv128LayerParam* bconv6 = new Conv128LayerParam("Conv6", bconv5->output_height, 
            bconv5->output_width, filter_height, filter_width, 512, 512, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv6_gpu = bconv6->initialize(config_file, bconv5->get_output_gpu());
    //Bconv7 Layer
    Conv128LayerParam* bconv7 = new Conv128LayerParam("Conv7", bconv6->output_height, 
            bconv6->output_width, filter_height, filter_width, 512, 512, batch, 1, 1,
            true, 2, 2, false,
            false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv7_gpu = bconv7->initialize(config_file, bconv6->get_output_gpu());
   
   
    //Bconv8 Layer
    Conv128LayerParam* bconv8 = new Conv128LayerParam("Conv8", bconv7->output_height, 
            bconv7->output_width, filter_height, filter_width, 512, 512, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv8_gpu = bconv8->initialize(config_file, bconv7->get_output_gpu());
    //Bconv9 Layer
    Conv128LayerParam* bconv9 = new Conv128LayerParam("Conv9", bconv8->output_height, 
            bconv8->output_width, filter_height, filter_width, 512, 512, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv9_gpu = bconv9->initialize(config_file, bconv8->get_output_gpu());
    //Bconv10 Layer
    Conv128LayerParam* bconv10 = new Conv128LayerParam("Conv10", bconv9->output_height, 
            bconv9->output_width, filter_height, filter_width, 512, 512, batch, 1, 1,
            true, 2, 2, false,
            false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv10_gpu = bconv10->initialize(config_file, bconv9->get_output_gpu());
    
    //Fc1 Layer
    Fc128LayerParam* bfc1 = new Fc128LayerParam("Fc1", batch, (bconv10->output_height)
            *(bconv10->output_width)*512, n_hidden, a_bit, w_bit); 
    Fc128LayerParam* bfc1_gpu = bfc1->initialize(config_file, bconv10->get_output_gpu());
    //Fc2 Layer
    Fc128LayerParam* bfc2 = new Fc128LayerParam("Fc2", batch, n_hidden, n_hidden, a_bit, w_bit); 
    Fc128LayerParam* bfc2_gpu = bfc2->initialize(config_file, bfc1->get_output_gpu());
    //Out Layer
    Out128LayerParam* bout = new Out128LayerParam("Fout", batch, n_hidden, output_size, a_bit, w_bit);
    Out128LayerParam* bout_gpu = bout->initialize(config_file, bfc2->get_output_gpu());  

    //================ Setup Kernel =================
    int numThreads = 512;
    int numBlocks = 16;
    int shared_memory = 65536; // 64KB

    hipFuncSetAttribute(reinterpret_cast<const void*>(Conv_new_global), hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory);
    hipFuncSetAttribute(reinterpret_cast<const void*>(FC_new_global), hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory);
    hipFuncSetAttribute(reinterpret_cast<const void*>(Output_new_global), hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory);

    std::clock_t c_start = std::clock();

    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv1_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv2_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv3_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv4_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv5_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv6_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv7_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv8_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv9_gpu);
    hipDeviceSynchronize(); 
    Conv_new_global<<<numBlocks, numThreads, shared_memory>>>(bconv10_gpu);
    hipDeviceSynchronize(); 
    FC_new_global<<<numBlocks, numThreads, shared_memory>>>(bfc1_gpu);
    hipDeviceSynchronize(); 
    FC_new_global<<<numBlocks, numThreads, shared_memory>>>(bfc2_gpu);
    hipDeviceSynchronize(); 
    Output_new_global<<<numBlocks, numThreads, shared_memory>>>(bout_gpu);
    hipDeviceSynchronize(); 

    hipError_t err = hipGetLastError();

    std::clock_t c_end = std::clock();
    float time_elapsed_ms = 1000.0f * (c_end-c_start) / CLOCKS_PER_SEC;
    printf("\n==============\nVGG (ms): %.3f\n", time_elapsed_ms);

    delete bconv1;
    delete bconv2;
    delete bconv3;
    delete bconv4;
    delete bconv5;
    delete bconv6;
    delete bconv7;
    delete bconv8;
    delete bconv9;
    delete bconv10;
    delete bfc1;
    delete bfc2;
    delete bout;

    return 0;

}