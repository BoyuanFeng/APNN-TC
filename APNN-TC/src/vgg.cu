#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------
// File: alexnet.cu
// VGG-16 BNN inference source file for ImageNet. 
// ---------------------------------------------------------------------------
// See our arXiv paper for detail: https://arxiv.org/abs/2006.16578
// Ang Li, Scientist, Pacific Northwest National Laboratory(PNNL), U.S.
// Homepage: http://www.angliphd.com
// GitHub repo: http://www.github.com/pnnl/TCBNN
// PNNL-IPID: 31925-E, ECCN: EAR99, IR: PNNL-SA-152850
// BSD Lincese.
// Richland, 99352, WA, USA. June-30-2020.
// ---------------------------------------------------------------------------

#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <iostream>
#include <string>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <fstream>
#include <vector>
#include "utility.h"
#include "param.h"
#include "kernel.cuh"
#include "data.h"

using namespace cooperative_groups;
using namespace std;

__global__ void vggnet128(
        InConv128LayerParam* bconv1, 
        Conv128LayerParam* bconv2, 
        Conv128LayerParam* bconv3,
        Conv128LayerParam* bconv4, 
        Conv128LayerParam* bconv5, 
        Conv128LayerParam* bconv6,
        Conv128LayerParam* bconv7, 
        Conv128LayerParam* bconv8,
        Conv128LayerParam* bconv9, 
        Conv128LayerParam* bconv10, 
        Conv128LayerParam* bconv11,
        Conv128LayerParam* bconv12,
        Conv128LayerParam* bconv13,
        Fc128LayerParam* bfc1, 
        Fc128LayerParam* bfc2, 
        Out128LayerParam* bout)
{
    grid_group grid = this_grid();
    
    //========= Conv1 ============
    InConv128Layer(bconv1);
    grid.sync();
    //========= Conv2 ============
    // Conv128Layer(bconv2);
    Conv_new(bconv2);
    grid.sync();
    //========= Conv3 ============
    // Conv128Layer(bconv3);
    Conv_new(bconv3);
    grid.sync();
    //========= Conv4 ============
    // Conv128Layer(bconv4);
    Conv_new(bconv4);
    grid.sync();
    //========= Conv5 ============
    // Conv128Layer(bconv5);
    Conv_new(bconv5);
    grid.sync();
    //========= Conv6 ============
    // Conv128Layer(bconv6);
    Conv_new(bconv6);
    grid.sync();
    //========= Conv7 ============
    // Conv128Layer(bconv7);
    Conv_new(bconv7);
    grid.sync();
    //========= Conv8 ============
    // Conv128Layer(bconv8);
    Conv_new(bconv8);
    grid.sync();
    //========= Conv9 ============
    // Conv128Layer(bconv9);
    Conv_new(bconv9);
    grid.sync();
    //========= Conv10 ============
    // Conv128Layer(bconv10);
    Conv_new(bconv10);
    grid.sync();
    //========= Conv11 ============
    // Conv128Layer(bconv11);
    Conv_new(bconv11);
    grid.sync();
    //========= Conv12 ============
    // Conv128Layer(bconv12);
    Conv_new(bconv12);
    grid.sync();
    //========= Conv13 ============
    // Conv128Layer(bconv13);
    Conv_new(bconv13);
    grid.sync();
    //========= Fc1 ============
//     Fc128Layer(bfc1);
    FC_new(bfc1);
    grid.sync();
    //========= Fc2 ============
//     Fc128Layer(bfc2);
    FC_new(bfc2);
    grid.sync();
    ////========== Output ===========
    // Out128Layer(bout);
    Output_new(bout);
}
  
     
int main()
{
    int dev = 0;
    hipSetDevice(dev);

    const unsigned batch = 8;
    const unsigned output_size = 1000;
    const unsigned image_height = 224;
    const unsigned image_width = 224;
    const unsigned image_channel = 3;
    const unsigned filter_height = 3;
    const unsigned filter_width = 3;
    const unsigned n_hidden = 4096;

    //=============== Get Input and Label =================
    float* images = (float*)malloc(batch*image_height*image_width*image_channel*sizeof(float));
    unsigned* image_labels = (unsigned*)malloc(batch*sizeof(unsigned));
//     read_ImageNet_normalized("./imagenet_files.txt", images, image_labels, batch);
    
    //================ Get Weight =================
    FILE* config_file = fopen("./vgg_imagenet.csv","r");
    //================ Set Network =================
    //Bconv1 Layer
    // InConv128LayerParam* bconv1 = new InConv128LayerParam("Conv1", image_height, image_width, 
    //         filter_height, filter_width, 3, 64, batch); 
    // InConv128LayerParam* bconv1_gpu = bconv1->initialize(images, config_file);

    uin32* lowBit_image_gpu = images_quantization(images, batch, image_height, image_width, image_channel);
    Conv128LayerParam* bconv1 = new Conv128LayerParam("Conv1", image_height, image_width, 
        filter_height, filter_width, 3, 64, batch); 
    Conv128LayerParam* bconv1_gpu = bconv1->initialize(config_file, lowBit_image_gpu);

    //Bconv2 Layer
    Conv128LayerParam* bconv2 = new Conv128LayerParam("Conv2", bconv1->output_height, 
            bconv1->output_width, filter_height, filter_width, 64, 64, batch, 1, 1,
            true, 2, 2, false, 
            false, false, 0, false, a_bit, w_bit
        );
        
    Conv128LayerParam* bconv2_gpu = bconv2->initialize(config_file, bconv1->get_output_gpu());
    //Bconv3 Layer
    Conv128LayerParam* bconv3 = new Conv128LayerParam("Conv3", bconv2->output_height, 
            bconv2->output_width, filter_height, filter_width, 64, 128, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv3_gpu = bconv3->initialize(config_file, bconv2->get_output_gpu());
    //Bconv4 Layer
    Conv128LayerParam* bconv4 = new Conv128LayerParam("Conv4", bconv3->output_height, 
            bconv3->output_width, filter_height, filter_width, 128, 128, batch, 1, 1,
            true, 2, 2, false,
            false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv4_gpu = bconv4->initialize(config_file, bconv3->get_output_gpu());
    //Bconv5 Layer
    Conv128LayerParam* bconv5 = new Conv128LayerParam("Conv5", bconv4->output_height, 
            bconv4->output_width, filter_height, filter_width, 128, 256, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv5_gpu = bconv5->initialize(config_file, bconv4->get_output_gpu());
    //Bconv6 Layer
    Conv128LayerParam* bconv6 = new Conv128LayerParam("Conv6", bconv5->output_height, 
            bconv5->output_width, filter_height, filter_width, 256, 256, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv6_gpu = bconv6->initialize(config_file, bconv5->get_output_gpu());
    //Bconv7 Layer
    Conv128LayerParam* bconv7 = new Conv128LayerParam("Conv7", bconv6->output_height, 
            bconv6->output_width, filter_height, filter_width, 256, 256, batch, 1, 1,
            true, 2, 2, false,
            false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv7_gpu = bconv7->initialize(config_file, bconv6->get_output_gpu());
    //Bconv8 Layer
    Conv128LayerParam* bconv8 = new Conv128LayerParam("Conv8", bconv7->output_height, 
            bconv7->output_width, filter_height, filter_width, 256, 512, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv8_gpu = bconv8->initialize(config_file, bconv7->get_output_gpu());
    //Bconv9 Layer
    Conv128LayerParam* bconv9 = new Conv128LayerParam("Conv9", bconv8->output_height, 
            bconv8->output_width, filter_height, filter_width, 512, 512, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv9_gpu = bconv9->initialize(config_file, bconv8->get_output_gpu());
    //Bconv10 Layer
    Conv128LayerParam* bconv10 = new Conv128LayerParam("Conv10", bconv9->output_height, 
            bconv9->output_width, filter_height, filter_width, 512, 512, batch, 1, 1,
            true, 2, 2, false,
            false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv10_gpu = bconv10->initialize(config_file, bconv9->get_output_gpu());
    //Bconv11 Layer
    Conv128LayerParam* bconv11 = new Conv128LayerParam("Conv11", bconv10->output_height, 
            bconv10->output_width, filter_height, filter_width, 512, 512, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv11_gpu = bconv11->initialize(config_file, bconv10->get_output_gpu());
    //Bconv12 Layer
    Conv128LayerParam* bconv12 = new Conv128LayerParam("Conv12", bconv11->output_height, 
            bconv11->output_width, filter_height, filter_width, 512, 512, batch,
            1, 1, true, 1, 1, false, false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv12_gpu = bconv12->initialize(config_file, bconv11->get_output_gpu());
    //Bconv13 Layer
    Conv128LayerParam* bconv13 = new Conv128LayerParam("Conv13", bconv12->output_height, 
            bconv12->output_width, filter_height, filter_width, 512, 512, batch, 1, 1,
            true, 2, 2, true,
            false, false, 0, false, a_bit, w_bit
        );
    Conv128LayerParam* bconv13_gpu = bconv13->initialize(config_file, bconv12->get_output_gpu());
    //Fc1 Layer
    Fc128LayerParam* bfc1 = new Fc128LayerParam("Fc1", batch, (bconv13->output_height)
            *(bconv13->output_width)*512, n_hidden, a_bit, w_bit); 
    Fc128LayerParam* bfc1_gpu = bfc1->initialize(config_file, bconv13->get_output_gpu());
    //Fc2 Layer
    Fc128LayerParam* bfc2 = new Fc128LayerParam("Fc2", batch, n_hidden, n_hidden, a_bit, w_bit); 
    Fc128LayerParam* bfc2_gpu = bfc2->initialize(config_file, bfc1->get_output_gpu());
    //Out Layer
    Out128LayerParam* bout = new Out128LayerParam("Fout", batch, n_hidden, output_size, a_bit, w_bit);
    Out128LayerParam* bout_gpu = bout->initialize(config_file, bfc2->get_output_gpu());  

    //================ Setup Kernel =================
    int numThreads = 256;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    int numBlocksPerSm;
    int shared_memory= 84*1e3; //84 KB

    hipFuncSetAttribute(reinterpret_cast<const void*>(vggnet128), hipFuncAttributeMaxDynamicSharedMemorySize,shared_memory);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, vggnet128, numThreads, shared_memory);

    void* args[] = {&bconv1_gpu, &bconv2_gpu, &bconv3_gpu, &bconv4_gpu, &bconv5_gpu, &bconv6_gpu,
        &bconv7_gpu, &bconv8_gpu, &bconv9_gpu, &bconv10_gpu, &bconv11_gpu, &bconv12_gpu, &bconv13_gpu,
        &bfc1_gpu, &bfc2_gpu, &bout_gpu};

    // shared_memory = 84 * 1e3; // 96KB
    printf("numBlocks: %d, shared_memory (KB): %.3f\n", numBlocksPerSm, 1.0f*shared_memory/1e3);
    START_TIMER;

    hipLaunchCooperativeKernel((void*)vggnet128, numBlocksPerSm*deviceProp.multiProcessorCount, 
            numThreads, args, shared_memory);

    STOP_TIMER;
    printf("VGG_b%d (ms): %.3f\n", batch, milliseconds);

    // float* output = bout->download_output();
    //validate_prediction(output, image_labels, output_size, batch);

    /*
    float* out = bconv2->download_full_output();
    for (int i=0; i<512; i++)
    //for (int i=4096; i<4096+512; i++)
    {
        printf("%.f ", out[i]);
        if ((i+1)%32==0) printf("\n");
    }
    printf("\n===%f===\n", bout->bn_scale[0]);
    */

    delete bconv1;
    delete bconv2;
    delete bconv3;
    delete bconv4;
    delete bconv5;
    delete bconv6;
    delete bconv7;
    delete bconv8;
    delete bconv9;
    delete bconv10;
    delete bconv11;
    delete bconv12;
    delete bconv13;
    delete bfc1;
    delete bfc2;
    delete bout;

    return 0;

}