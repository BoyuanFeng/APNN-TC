#include "hip/hip_runtime.h"
/*
  1-bit BMMA code.
  Runs at 500TOPS for matrix size of 4096x4096x8192.
  Borrows largely from CUDA-SDK.

  By Boyuan
*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define CHUNK_K 4
#define SKEW 1
#define WARPS_PER_BLOCK 8
#define WARP_SIZE 32
#define THREADS_PER_BLOCK WARP_SIZE * WARPS_PER_BLOCK
#define CHUNK_LINE_BYTES CHUNK_K * sizeof(int4)
#define WARP_COPY_BYTES WARP_SIZE * sizeof(int4)
#define CHUNK_COPY_LINES_PER_WARP WARP_COPY_BYTES / CHUNK_LINE_BYTES
#define CHUNK_COPY_LINE_LANES WARP_SIZE / CHUNK_COPY_LINES_PER_WARP
#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4
#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2
#define BLOCK_ROW_TILES WARP_ROW_TILES * BLOCK_ROW_WARPS
#define BLOCK_COL_TILES WARP_COL_TILES * BLOCK_COL_WARPS
#define M 8
#define N 8
#define K 128

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;
using namespace nvcuda::wmma::experimental;

typedef union {
  int4 vec;
  int a[4];
} U4;


// Assume that Kernel size is 3x3.
// Assume CIN is 128.
__global__ void compute_conv_imma(const int4 *W, const int4 *X, int *Output, int Height, int Width, int CIN, int COUT) {
  // GEMM Configuration
  int X_bit_offset = Height * Width * CIN/128;

  extern __shared__ int4 shmem[][CHUNK_K+SKEW]; // TODO: Padding opportunity may exist here.
  wmma::fragment<wmma::accumulator, 8, 8, 128, int> c[WARP_COL_TILES]
    [WARP_ROW_TILES];

  // Warp and lane identification.
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
    const unsigned int block_i = (block_pos/(COUT/64)) / (Width/8) * 4;
    const unsigned int block_j = (block_pos/(COUT/64)) % (Width/8) * 8;
    const unsigned int block_z = block_pos % (COUT/64) * 64;

    if (block_i >= Height) {
      break;
    }

    int image_starting_idx = block_i * 4 * Width * CIN/32 + block_j * 8 * CIN/32;

    for(int i=0; i < WARP_COL_TILES; i++)
      for(int j=0; j < WARP_ROW_TILES; j++)
        wmma::fill_fragment(c[i][j], 0);
    
    // Go through the global K dimension by a fixed step at a time.
#pragma unroll
    for (int tile_k = 0; tile_k < int(9*CIN/128/4); tile_k += CHUNK_K) {

      int SHMEM_i = threadIdx.x/4;
      int SHMEM_part = SHMEM_i / 32;
      int SHMEM_offset = SHMEM_i % 32;
      int row = SHMEM_offset / 8;
      int col = SHMEM_offset % 8;
      int t = threadIdx.x % 4;

      int GL_idx = image_starting_idx + SHMEM_part*X_bit_offset + row*Width*CIN/128 + col*CIN/18 + tile_k*4 + t;

      shmem[SHMEM_i][t] = X[GL_idx];

      SHMEM_i += 64;
      int weight_load_idx = SHMEM_part * 9 * CIN * COUT / 128 + (block_z + SHMEM_offset) * 9 * CIN/128;
      int thread_load_idx = weight_load_idx + (tile_k*4 + t) * CIN/128;
      shmem[SHMEM_i][t] = W[thread_load_idx];

      __syncthreads();

      // Compute a grid of C matrix tiles in each warp.
#pragma unroll
      for (int k_step = 0; k_step < CHUNK_K; k_step++) {
        wmma::fragment<wmma::matrix_a, M, N, K, precision::b1, wmma::row_major> a[WARP_COL_TILES];
        wmma::fragment<wmma::matrix_b, M, N, K, precision::b1, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
          size_t shmem_idx_a = (warpId / 2) * M * 4 + (i * M);
          const int4 *tile_ptr = &shmem[shmem_idx_a][k_step];

          wmma::load_matrix_sync(a[i], tile_ptr, (CHUNK_K + SKEW)*128);

#pragma unroll
          for (int j = 0; j < WARP_ROW_TILES; j++) {
            if (i == 0) {
              // Load the B matrix fragment once, because it is going to be
              // reused against the other A matrix fragments.
              size_t shmem_idx_b = 64 +
                                   (WARP_ROW_TILES * N) * (warpId % 2) +
                                   (j * N);
              const int4 *tile_ptr = &shmem[shmem_idx_b][k_step * (K/128)];

              wmma::load_matrix_sync(b[j], tile_ptr, (CHUNK_K + SKEW)*128);
            }
            // printf("ckpt4\n");

            wmma::bmma_sync(c[i][j], a[i], b[j], c[i][j]);
          }
        }
      }
      __syncthreads();
    }

    // Needs special handle for the remaining K.

    // Store the D fragments to shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
        int *tile_ptr = (int*)&shmem[0][0] + warpId*8*64 + (i*4+j) * 64;
        wmma::store_matrix_sync(tile_ptr, c[i][j], 8,  wmma::mem_row_major);
      }
    }

    __syncthreads();



    U4 tmp0;
    U4 tmp1;
    U4 tmp2;
    U4 tmp3;
    U4 val;

    int *shmem_warp_stream_ptr = (int*)&shmem[0][0]+threadIdx.x/8*64 + (threadIdx.x%8)*4;
    tmp0.vec = *((int4*)shmem_warp_stream_ptr);
    tmp1.vec = *((int4*)shmem_warp_stream_ptr+8);
    tmp2.vec = *((int4*)shmem_warp_stream_ptr+32*16);
    tmp3.vec = *((int4*)shmem_warp_stream_ptr+32*16+8);
    val.a[0] = tmp0.a[0] + 2*tmp1.a[0] + 2*tmp2.a[0] + 4*tmp3.a[0];
    val.a[1] = tmp0.a[1] + 2*tmp1.a[1] + 2*tmp2.a[1] + 4*tmp3.a[1];
    val.a[2] = tmp0.a[2] + 2*tmp1.a[2] + 2*tmp2.a[2] + 4*tmp3.a[2];
    val.a[3] = tmp0.a[3] + 2*tmp1.a[3] + 2*tmp2.a[3] + 4*tmp3.a[3];

    int shmem_row = threadIdx.x/8;
    int row = shmem_row / 8;
    int col = shmem_row % 8;
    int* dst_gmem_warp_stream_ptr = Output + block_i * 4 * Width * COUT + block_j*8*COUT 
              + row*Width*COUT + col*4;
    *(int4*)dst_gmem_warp_stream_ptr = val.vec;
    __syncthreads();
  }
}

// void init_matrices(int4 *A, int4 *B){
//   int *A_int = (int*) A;
//   int *B_int = (int*) B;
//   for(int i = 0; i < M_GLOBAL; i++) {
//     for(int j = 0; j < K_GLOBAL/32; j++) {
//       A_int[i*K_GLOBAL/32+j] = rand();
//     }
//   }

//   for(int i = 0; i < N_GLOBAL; i++) {
//     for(int j = 0; j < K_GLOBAL/32; j++) {
//       B_int[i*K_GLOBAL/32+j] = 0xFFFFFFFF;
//       B_int[i*K_GLOBAL/32+j] = rand();
//     }
//   }
// }

// int popcnt(int i) {
//      // Java: use int, and use >>> instead of >>
//      // C or C++: use int
//      i = i - ((i >> 1) & 0x55555555);
//      i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
//      return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
// }

// void compute_ref(int4 *A, int4 *B, int *ref_C) {
//   int *A_int = (int*) A;
//   int *B_int = (int*) B;

//   for (int m = 0; m < M_GLOBAL; m++) {
//     for (int n = 0; n < N_GLOBAL; n++) {
//       int tmp = 0;
//       for (int k = 0; k < K_GLOBAL; k += 32) {
//         // bit vector from row A and column B, accumulation and addition.
//         tmp += popcnt(A_int[(m*K_GLOBAL + k)/32] ^ B_int[(n*K_GLOBAL + k)/32]);
//       }
//       // ref_C[m * K + n]= K - 2 * tmp;
//       ref_C[m * N_GLOBAL + n]= tmp;
//     }
//   }
// }


// void validate_results(int *C, int* ref_C, int M_, int N_) {
//   printf("Checking computed result for correctness: ");
//   bool correct = true;
//   double eps = 1.e-6;  // machine zero

//   for(int i = 0; i < M_; i++) {
//     for(int j = 0; j < N_; j++) {
//       int idx = i*N_+j;
//       double dst = fabs(C[idx] - ref_C[idx]);
//       double abs = fabs(C[idx]) * fabs(ref_C[idx]);
//       double ref_err = dst / abs;
//       if (ref_err > eps) {
//         // printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",, eps);
//         printf("i: %d, j: %d, C: %d, ref_C: %d\n", i, j, C[idx], ref_C[idx]);
//         // printf("non equal\n");
//         correct = false;
//       }
//     }
//   }
//   printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");
// }

// #define verify_output

int main(int argc, char **argv) {
  printf("Initializing...\n");

  int dev = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

  int Height = 128;
  int Width = 16;
  int CIN = 128;
  int COUT = 256;
  int bit = 2;

  int4 *X = NULL;
  int4 *W = NULL;
  int *Output = NULL;

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&X), sizeof(int4) * Height * Width * (CIN/128) * bit));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&W), sizeof(int4) * 9 * (CIN/128) * COUT * bit));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&Output), sizeof(int4) * Height * Width * COUT ));

// #ifdef verify_output
//   printf("Preparing validation data for GPU...\n");
// int4 *W_h = NULL;
// int4 *X_h = NULL;
// int *Output_h = NULL;

// X_h = (int4 *)malloc(sizeof(int4) * H * W * (CIN/128) * X_bit);
// W_h = (int4 *)malloc(sizeof(int4) * 9 * (CIN/128) * COUT * W_bit);
// Output_h = (int *)malloc(sizeof(int4) * H * W * (COUT/128) * X_bit);
//   init_matrices(A_h, B_h);
//   checkCudaErrors(hipMemcpy(A, A_h, sizeof(int4) * M_GLOBAL * (K_GLOBAL/128), hipMemcpyHostToDevice));
//   checkCudaErrors(hipMemcpy(B, B_h, sizeof(int4) * N_GLOBAL * (K_GLOBAL/128), hipMemcpyHostToDevice));
// #endif

  int SHMEM_SZ = 65536;
  checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(
    compute_conv_imma), hipFuncAttributeMaxDynamicSharedMemorySize,
    SHMEM_SZ));

  // Run ours NUM_PROFILES times and record time.
  float bmma_ms_avg = 0.0f;
  int NUM_PROFILES = 1000;
  for(int iter=0; iter<NUM_PROFILES; ++iter){
          float bmma_ms = 0.0f;
          hipEvent_t bmma_start;
          hipEvent_t bmma_end;
          hipEventCreate(&bmma_start);
          hipEventCreate(&bmma_end);
          hipEventRecord(bmma_start);
          checkKernelErrors(
            (compute_conv_imma<<<deviceProp.multiProcessorCount, THREADS_PER_BLOCK,
                                  SHMEM_SZ>>>(W, X, Output, Height, Width, CIN, COUT)));
                hipEventRecord(bmma_end);
          hipEventSynchronize(bmma_end);
          hipEventElapsedTime(&bmma_ms, bmma_start, bmma_end);
          hipEventDestroy(bmma_start);
          hipEventDestroy(bmma_end);
          bmma_ms_avg += bmma_ms;
  }

  bmma_ms_avg = bmma_ms_avg/(double)NUM_PROFILES;

  printf("Time: %f ms\n", bmma_ms_avg);

  printf("TOPS: %.2f\n", (((double)9 * CIN * Height * Width * COUT * 2)/(bmma_ms_avg/1000.)) / 1e12);


// #ifdef verify_output
//   printf("Validating results...\n");
//   checkCudaErrors(hipMemcpy(C_h, C, sizeof(int) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost));

//   int *C_ref = (int *)malloc(sizeof(int) * M_GLOBAL * N_GLOBAL);

//   /* Copmpute reference matrix on CPU */
//   // compute_ref(A_h, B_h, C_ref);

//   /* validation results */
//   // validate_results(C_h, C_ref, M_GLOBAL, N_GLOBAL);
// #endif

  // free(A_h);
  // free(B_h);
  // free(C_h);
  // checkCudaErrors(hipFree(reinterpret_cast<void *>(A)));
  // checkCudaErrors(hipFree(reinterpret_cast<void *>(B)));
  // checkCudaErrors(hipFree(reinterpret_cast<void *>(C)));

  return EXIT_SUCCESS;
}
