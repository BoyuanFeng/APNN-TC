#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// CUDA sample demonstrating a integer GEMM computation using the Warp Matrix
// Multiply and Accumulate API.

// In this program, the compute_gemm kernel computes the result of a matrix
// multiplication and addition: D = alpha * A * B + beta * C. The dimensions of
// both C and D matrices are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x
// K_GLOBAL (row-major), the B matrix is K_GLOBAL x N_GLOBAL (column-major). In
// that kernel, each CTA computes one 128 x 128 tile of the resulting matrix per
// iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 128 x 128 tile to compute.
// Each CTA consists of eight warps. For the 128 x 128 tile, each warp computes
// eight 16 x 16 subtiles, organized in a 2 x 4 two-dimensional array. Warps
// compute the 16 x 16 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and
// accumulating the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 128 x 128 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments
//   from shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B
// matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the
//   A and B data from shared memory, thus reducing the number of data copies
//   from global memory.
// - The portions of the A and B matrices are stored in shared memory with an
// additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_HALF macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each
// warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory
//   contents to global memory, again avoiding redundant random global memory
//   accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register
// utilization,
//   but carefully enough to avoid local memory use.

#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Externally configurable parameters.

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 1
#endif

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 8
#define N 8
#define K 128

// GEMM configuration.

#define M_TILES 1024
#define N_TILES 1024
#define K_TILES 512

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that are 16 * 16 * 8 * 8 * 2 = 32 Kb each
// (i.e. two 8x8 arrays of tiles of 16x16 uint8_t-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the
// performance would be severely impacted. So we choose to reduce the chunk size
// in half, i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 8
#else
#define CHUNK_K 16
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * sizeof(int4))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B
// matrix in shared memory to minimize possible bank conflicts. Before
// performing the nvcuda::wmma::mma_sync operation, the warp must load the
// matrix data using the nvcuda::wmma::load_matrix_sync operation. Although the
// memory access pattern is not specified for that function, each lane in the
// warp can read one or multiple matrix elements from different matrix rows or
// columns. For shared memory, such access can result in bank conflicts if
// different rows / columns of the matrix map to the same bank. By shifting each
// row and column by a few bytes, we make sure that they map to different banks,
// thus reducing the number of possible bank conflicts. The number of 32
// one-byte "uint8_t" elements is chosen as the minimum possible shift because
// we must keep each row and column 256-bit aligned, as required by
// nvcuda::wmma::load_matrix_sync.
#define SKEW 2 // Updated for int4

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;
using namespace nvcuda::wmma::experimental;

__global__ void compute_gemm_imma(const int4 *A, const int4 *B, int *D) {
  extern __shared__ int4 shmem[][CHUNK_K+SKEW]; // TODO: Padding opportunity may exist here.

  // Warp and lane identification.
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  // Each CTA slides along the 128 x 128 tiles from the top left corner of the
  // matrix to the right and down, and selects the next tile to compute. Once
  // there's no such tile, all warps in this CTA exit.
  for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
    const unsigned int block_tile_i =
        (block_pos * BLOCK_ROW_TILES * BLOCK_COL_TILES) / N_TILES;
    const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

    // Stop when there are no more D matrix tiles to compute in this CTA.
    if (block_tile_i >= M_TILES) {
      break;
    }

    // These fragments will accumulate the result of A and B matrix fragment
    // multiplications along the K_GLOBAL dimension.
    wmma::fragment<wmma::accumulator, M, N, K, int> c[WARP_COL_TILES]
                                                     [WARP_ROW_TILES];

    for(int i=0; i < WARP_COL_TILES; i++)
      for(int j = 0; j < WARP_ROW_TILES; j++)
        wmma::fill_fragment(c[i][j], 0);

    // warpId = 4;
    // printf("warpId == 4: %d", (warpId == 4));


    // if ((warpId == (unsigned int)4) && (laneId > 0)) {
    //   printf("warpId: %d, laneId: %d. ckpt1\n", warpId, laneId);
    // }

    
    // Select what warp copies what matrix to shared memory.
    // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
    const int4 *warp_ptr = (warpId < 4) ? (&A[block_tile_i * M * (K_GLOBAL/128)] +
                                              M * (K_GLOBAL/128) * (warpId % 4) * 2)
                                           : (&B[block_tile_j * N * (K_GLOBAL/128)] +
                                              N * (K_GLOBAL/128) * (warpId % 4) * 2);

    // Go through the global K dimension by a fixed step at a time.
#pragma unroll
    for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
      // Offset in shared memory from which the B matrix is stored.
      const size_t shmem_idx_b_off = BLOCK_COL_TILES * M; // TODO: This BLOCK_COL_TILES may be selected to improve performance. Maybe moved outside the for loop.

      // printf("shmem_idx_b_off: %d\n", shmem_idx_b_off);

      // Copy slices of the A and B matrices to shared memory.
      // The first half of the warps in the CTA copy the A matrix, the rest copy
      // the B matrix.
      size_t shmem_idx =
          warpId < (WARPS_PER_BLOCK / 2)
              ? (M * (warpId % (WARPS_PER_BLOCK / 2)) * 2)
              : (N * (warpId % (WARPS_PER_BLOCK / 2)) * 2 + shmem_idx_b_off);

      // if(shmem_idx > 100) {
        // printf("Line 237: shmem_idx: %d\n", shmem_idx);
      // }

      // First half of the warp copies the first row / column of the matrix,
      // the second half of the warp copies the next.
      int4 *lane_ptr = (int4 *)(warp_ptr + tile_k * (K/128) +
                                (laneId / CHUNK_COPY_LINE_LANES) * (K_GLOBAL/128)) +
                       (laneId % CHUNK_COPY_LINE_LANES); // (K/128), since K=128 in bit. int4 is 128 bit.
                       
      // Shift the second half of the warp to the next row / column in the
      // shared memory.
      shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

      // if(shmem_idx > 100) {
      //   printf("Line 251: shmem_idx: %d\n", shmem_idx);
      // }


#pragma unroll
      for (int i = 0; i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP); i++) {
        unsigned int tmp = CHUNK_COPY_LINE_LANES;
        // unsigned int tmp1 = laneId % tmp;
        // printf("laneId: %u, Zero: %d\n", laneId, 5+6);
        // if(shmem_idx > 100 and laneId == 0) {
        //   printf("laneId: %u, (laneId mod CHUNK_COPY_LINE_LANES): %d, shmem_idx: %d\n", laneId, laneId % tmp, shmem_idx);
        // }
        // Copy 16 bytes at once in each lane.
        *((int4 *)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) =
            *lane_ptr;

        // Advance the global memory pointer and the shared memory index.
        lane_ptr = (int4 *)(lane_ptr +
                            (K_GLOBAL/128) * CHUNK_COPY_LINES_PER_WARP);
        shmem_idx += CHUNK_COPY_LINES_PER_WARP;
      }

      __syncthreads();


      // if (warpId == 0 && laneId == 0) {
      //   printf("ckpt2\n");
      // }
  
      // Compute a grid of C matrix tiles in each warp.
#pragma unroll
      for (int k_step = 0; k_step < CHUNK_K; k_step++) {
        wmma::fragment<wmma::matrix_a, M, N, K, precision::b1, wmma::row_major> a[WARP_COL_TILES];
        wmma::fragment<wmma::matrix_b, M, N, K, precision::b1, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
          size_t shmem_idx_a = (warpId / 2) * M * 2 + (i * M);
          const int4 *tile_ptr = &shmem[shmem_idx_a][k_step * (K/128)];

          wmma::load_matrix_sync(a[i], tile_ptr, CHUNK_K + SKEW);

          // printf("ckpt3\n");

#pragma unroll
          for (int j = 0; j < WARP_ROW_TILES; j++) {
            if (i == 0) {
              // Load the B matrix fragment once, because it is going to be
              // reused against the other A matrix fragments.
              size_t shmem_idx_b = shmem_idx_b_off +
                                   (WARP_ROW_TILES * N) * (warpId % 2) +
                                   (j * N);
              const int4 *tile_ptr = &shmem[shmem_idx_b][k_step * (K/128)];

              wmma::load_matrix_sync(b[j], tile_ptr, CHUNK_K + SKEW);
            }
            // printf("ckpt4\n");

            wmma::bmma_sync(c[i][j], a[i], b[j], c[i][j]);
          }
        }
      }
      __syncthreads();
    }
    
    // 接下来还需要认真修改一下。现在write-to-GL多花了一倍的memory access.

    // This pointer is used to access the C and D matrix tiles this warp computes.
    int4 *shmem_warp_tile_ptr = &shmem[0][0] +
                              (warpId / 2) * SHMEM_STRIDE * (K/128) * 2 +
                              (warpId % 2) * SHMEM_OFFSET; // Will be used only when writing back D. May be moved outside the for loop. TODO.

      // Store the D fragments to shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
        int *tile_ptr = (int*)shmem_warp_tile_ptr + i * SHMEM_STRIDE * (K/64) + j * N; // Problem here.

        // int4 *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * (K/128) + j * N;

        wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
      }
    }

    __syncthreads();

    // This pointer is used to stream the C and D matrices block-wide tile to and
    // from shared memory.
    int4 *shmem_warp_stream_ptr = &shmem[0][0] + warpId * SHMEM_STRIDE * (K/128); // Will be used only when writing back D. Maybe moved outside the for loop. TODO.

    // This warp's pointer to the C matrix data to copy memory from to shared memory. 
    // TODO: May be moved outside the for loop.
    const size_t gmem_idx =
        (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;

        // Now that shared memory contains all the D tiles, stream them to global
    // memory.
    int *dst_gmem_warp_stream_ptr = (int *)&D[gmem_idx];

#pragma unroll
    for (int i = 0; i < K; i++) {
      *((int4 *)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
      *((int4 *)(shmem_warp_stream_ptr) + laneId);
          // *((int4 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
    }

    __syncthreads();
  }
}

int main(int argc, char **argv) {
  printf("Initializing...\n");

  int dev = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

  printf("M: %d (%d x %d)\n", M_GLOBAL, M, M_TILES);
  printf("N: %d (%d x %d)\n", N_GLOBAL, N, N_TILES);
  printf("K: %d (%d x %d)\n", K_GLOBAL, K, K_TILES);

  int4 *A_h = NULL;
  int4 *B_h = NULL;
  int *C_h = NULL;

  A_h = (int4 *)malloc(sizeof(int4) * M_GLOBAL * (K_GLOBAL/128));
  B_h = (int4 *)malloc(sizeof(int4) * (K_GLOBAL/128) * N_GLOBAL);
  C_h = (int *)malloc(sizeof(int) * M_GLOBAL * N_GLOBAL);

  int4 *A = NULL;
  int4 *B = NULL;
  int *C = NULL;

  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&A), sizeof(int4) * M_GLOBAL * (K_GLOBAL/128)));
  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&B), sizeof(int4) * N_GLOBAL * (K_GLOBAL/128)));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&C), sizeof(int) * M_GLOBAL * N_GLOBAL));

  assert(((unsigned long long)A) % 128 == 0);
  assert(((unsigned long long)B) % 128 == 0);
  assert(((unsigned long long)C) % 128 == 0);

  enum {
    // Compute the right amount of shared memory to request.
    // We need shared memory to hold per-CTA C and D matrix tiles, and to cache
    // per-CTA chunks
    // of the A and B matrices. Therefore, the right amount to request is the
    // maximum of those
    // two numbers.
    SHMEM_SZ = MAX(sizeof(int4) * (BLOCK_COL_TILES * M) *
                       (CHUNK_K * (K/128) + SKEW) * 2,
                   M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N *
                       (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(int))
  };

  printf("Required shared memory size: %lu Kb\n", SHMEM_SZ / 1024UL);

  checkCudaErrors(hipMemcpy(A, A_h, sizeof(int4) * M_GLOBAL * (K_GLOBAL/128),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(B, B_h, sizeof(int4) * N_GLOBAL * (K_GLOBAL/128),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(C, C_h, sizeof(int) * M_GLOBAL * N_GLOBAL,
                             hipMemcpyHostToDevice));

  printf("Preparing data for GPU...\n");

  assert(((unsigned long long)A) % 128 == 0);
  assert(((unsigned long long)B) % 128 == 0);
  assert(((unsigned long long)C) % 128 == 0);

  hipEvent_t start, stop;

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start));

  // If enough shared memory available on the GPU use high performant kernel
  printf("Computing... using high performance kernel compute_gemm_imma \n");

  checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(
      compute_gemm_imma), hipFuncAttributeMaxDynamicSharedMemorySize,
      SHMEM_SZ));
  checkKernelErrors(
      (compute_gemm_imma<<<deviceProp.multiProcessorCount, THREADS_PER_BLOCK,
                            SHMEM_SZ>>>(A, B, C)));


  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));

  float milliseconds = 0;

  checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    printf("Time: %f ms\n", milliseconds);
    printf("TOPS: %.2f\n", (((double)M_GLOBAL * N_GLOBAL * K_GLOBAL * 2)/(milliseconds/1000.)) / 1e12);

  free(A_h);
  free(B_h);
  free(C_h);
  checkCudaErrors(hipFree(reinterpret_cast<void *>(A)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(B)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(C)));

  return EXIT_SUCCESS;
}
